
#include <hip/hip_runtime.h>

/*
Ifrit-v2
Copyright (C) 2024 funkybirds(Aeroraven)

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU Affero General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU Affero General Public License for more details.

You should have received a copy of the GNU Affero General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>. */


#ifdef IFRIT_FEATURE_CUDA
#include "DefaultDemoShaders.cuh"
#include "core/cuda/CudaUtils.cuh"
#include "engine/math/ShaderBuiltinCuda.cuh"

namespace Ifrit::Demo::DemoDefault {
	IFRIT_DUAL void DemoVertexShaderCuda::execute(const void* const* input, ifloat4* outPos, ifloat4* const* outVaryings) {
		using namespace Ifrit::SoftRenderer::Math::ShaderOps::CUDA;
		//float4x4 view = (lookAt({ 0,1.5,5.25 }, { 0,1.5,0.0 }, { 0,1,0 }));
		//float4x4 view = (lookAt({ 0,0.75,1.50 }, { 0,0.75,0.0 }, { 0,1,0 }));
		//float4x4 view = (lookAt({ 0,0.1,1.25 }, { 0,0.1,0.0 }, { 0,1,0 }));
		float4x4 view = (lookAt({ 0.08,0.1,0.25 }, { 0,0.1,0.0 }, { 0,1,0 }));  //fox
		//float4x4 view = (lookAt({ 0.0,0.6,-1.5 }, { 0,0.4,0.0 }, { 0,1,0 }));  //af 
		 
		//float4x4 view = (lookAt({ 0,0.1,0.25 }, { 0,0.1,0.0 }, { 0,1,0 }));
		//float4x4 view = (lookAt({ 500,300,0 }, { -100,300,-0 }, { 0,1,0 }));
		//float4x4 proj = (perspective(60 * 3.14159 / 180, 1920.0 / 1080.0, 10.0, 3000));

		//float4x4 view = (lookAt({ 0,1.5,0}, { -100,1.5,0 }, { 0,1,0 }));
		float4x4 proj = (perspective(60 * 3.14159 / 180, 1920.0 / 1080.0, 0.1, 1000));
		float4x4 mvp = multiply(proj, view);
		auto s = isbReadFloat4(input[0]);
		auto p = multiply(mvp, s);
		*outPos = p;
		*outVaryings[0] = isbReadFloat4(input[1]);
		*outVaryings[1] = isbReadFloat4(input[2]);
		(*outVaryings[1]).y = 1.0f - outVaryings[1]->y;
	}

	IFRIT_HOST Ifrit::SoftRenderer::VertexShader* DemoVertexShaderCuda::getCudaClone() {
		return Ifrit::SoftRenderer::Core::CUDA::hostGetDeviceObjectCopy<DemoVertexShaderCuda>(this);
	}

	IFRIT_DUAL void DemoFragmentShaderCuda::execute(const  void* varyings, void* colorOutput, float* fragmentDepth) {
		using Ifrit::SoftRenderer::Math::ShaderOps::CUDA::abs;
		using Ifrit::SoftRenderer::Math::ShaderOps::CUDA::texture;
		using Ifrit::SoftRenderer::Math::ShaderOps::CUDA::textureLod;

		auto result = isbcuReadPsVarying(varyings, 0);
		auto& co = isbcuReadPsColorOut(colorOutput, 0);
		//auto dco = isbcuSampleTexLod(0, 0, float2( result.x, 1.0f - result.y ),2.5f); 
		//auto dcl = static_cast<const ifloat4s256*>(varyings);
		//float2 uv = { dcl[1].x,dcl[1].y };
		//auto dco = texture(0, 0, dcl, 1);

		co.x = result.x * 0.5 + 0.5;
		co.y = result.y * 0.5 + 0.5;
		co.z = result.z * 0.5 + 0.5;
		co.w = 0.5;

		/*
		co.x = result.x;
		co.y = result.y;
		co.z = result.z;
		co.w = 0.5;*/
		//printf("%f %f %f %f\n", result.x, result.y, result.z, result.w);
	}

	IFRIT_HOST Ifrit::SoftRenderer::FragmentShader* DemoFragmentShaderCuda::getCudaClone() {
		return Ifrit::SoftRenderer::Core::CUDA::hostGetDeviceObjectCopy<DemoFragmentShaderCuda>(this);
	}

	IFRIT_DUAL void DemoGeometryShaderCuda::execute(const ifloat4* const* inPos, const Ifrit::SoftRenderer::VaryingStore* const* inVaryings,
		ifloat4* outPos, Ifrit::SoftRenderer::VaryingStore* outVaryings, int* outSize) {
		outPos[0] = *inPos[0];
		outPos[1] = *inPos[1];
		outPos[2] = *inPos[2];

		outPos[0].x += 0.03;
		outPos[1].x += 0.03;
		outPos[2].x += 0.03;

		isbStoreGsVarying(0, 0, 2, isbReadGsVarying(0, 0));
		isbStoreGsVarying(0, 1, 2, isbReadGsVarying(0, 1));
		isbStoreGsVarying(1, 0, 2, isbReadGsVarying(1, 0));
		isbStoreGsVarying(1, 1, 2, isbReadGsVarying(1, 1));
		isbStoreGsVarying(2, 0, 2, isbReadGsVarying(2, 0));
		isbStoreGsVarying(2, 1, 2, isbReadGsVarying(2, 1));
		*outSize = 3;
	}

	IFRIT_HOST Ifrit::SoftRenderer::GeometryShader* DemoGeometryShaderCuda::getCudaClone() {
		return  Ifrit::SoftRenderer::Core::CUDA::hostGetDeviceObjectCopy<DemoGeometryShaderCuda>(this);
	}
}
#endif