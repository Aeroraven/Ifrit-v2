#include "hip/hip_runtime.h"
#include "engine/tilerastercuda/TileRasterInvocationCuda.cuh"
#include "engine/math/ShaderOpsCuda.cuh"
#include "engine/tilerastercuda/TileRasterDeviceContextCuda.cuh"
#include "engine/tilerastercuda/TileRasterConstantsCuda.h"
namespace Ifrit::Engine::TileRaster::CUDA::Invocation::Impl {
	IFRIT_DEVICE_CONST static int csFrameWidth = 0;
	IFRIT_DEVICE_CONST static int csFrameHeight = 0;
	IFRIT_DEVICE_CONST static bool csCounterClosewiseCull = false;
	IFRIT_DEVICE_CONST static int csVertexOffsets[CU_MAX_ATTRIBUTES];
	IFRIT_DEVICE_CONST static int csTotalVertexOffsets = 0;

	static int hsFrameWidth = 0;
	static int hsFrameHeight = 0;
	static bool hsCounterClosewiseCull = false;
	static int hsVertexOffsets[CU_MAX_ATTRIBUTES];
	static int hsTotalVertexOffsets = 0;

	IFRIT_DEVICE float devEdgeFunction(ifloat4 a, ifloat4 b, ifloat4 c) {
		return (c.x - a.x) * (b.y - a.y) - (c.y - a.y) * (b.x - a.x);
	}
	IFRIT_DEVICE bool devTriangleCull(ifloat4 v1, ifloat4 v2, ifloat4 v3) {
		float d1 = (v1.x * v2.y);
		float d2 = (v2.x * v3.y);
		float d3 = (v3.x * v1.y);
		float n1 = (v3.x * v2.y);
		float n2 = (v1.x * v3.y);
		float n3 = (v2.x * v1.y);
		float d = d1 + d2 + d3 - n1 - n2 - n3;
		if (d < 0.0f) return false;
		return true;
	}

	IFRIT_DEVICE void devGetAcceptRejectCoords(ifloat3 edgeCoefs[3], int chosenCoordTR[3], int chosenCoordTA[3]) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;
		for (int i = 0; i < 3; i++) {
			bool normalRight = edgeCoefs[i].x < 0;
			bool normalDown = edgeCoefs[i].y < 0;
			if (normalRight) {
				if (normalDown) {
					chosenCoordTR[i] = VRB;
					chosenCoordTA[i] = VLT;
				}
				else {
					chosenCoordTR[i] = VRT;
					chosenCoordTA[i] = VLB;
				}
			}
			else {
				if (normalDown) {
					chosenCoordTR[i] = VLB;
					chosenCoordTA[i] = VRT;
				}
				else {
					chosenCoordTR[i] = VLT;
					chosenCoordTA[i] = VRB;
				}
			}
		}
	}

	IFRIT_DEVICE bool devTriangleSimpleClip(ifloat4 v1, ifloat4 v2, ifloat4 v3, irect2Df& bbox) {
		bool inside = true;
		float minx = min(v1.x, min(v2.x, v3.x));
		float miny = min(v1.y, min(v2.y, v3.y));
		float maxx = max(v1.x, max(v2.x, v3.x));
		float maxy = max(v1.y, max(v2.y, v3.y));
		float maxz = max(v1.z, max(v2.z, v3.z));
		float minz = min(v1.z, min(v2.z, v3.z));
		if (maxz < 0.0f) return false;
		if (minz > 1.0f) return false;
		if (maxx < 0.0f) return false;
		if (minx > 1.0f) return false;
		if (maxy < 0.0f) return false;
		if (miny > 1.0f) return false;
		bbox.x = minx;
		bbox.y = miny;
		bbox.w = maxx - minx;
		bbox.h = maxy - miny;
		return true;
	}
	IFRIT_DEVICE void devExecuteBinner(
		int primitiveId,
		AssembledTriangleProposalCUDA& atp,
		irect2Df bbox,
		uint32_t** dRasterQueue,
		uint32_t* dRasterQueueCount,
		TileBinProposalCUDA** dCoverQueue,
		uint32_t* dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;
		float minx = bbox.x ;
		float miny = bbox.y ;
		float maxx = (bbox.x + bbox.w);
		float maxy = (bbox.y + bbox.h);

		int tileMinx = max(0, (int)(minx * CU_TILE_SIZE));
		int tileMiny = max(0, (int)(miny * CU_TILE_SIZE));
		int tileMaxx = min(CU_TILE_SIZE - 1, (int)(maxx * CU_TILE_SIZE));
		int tileMaxy = min(CU_TILE_SIZE - 1, (int)(maxy * CU_TILE_SIZE));

		ifloat3 edgeCoefs[3];
		edgeCoefs[0] = atp.e1;
		edgeCoefs[1] = atp.e2;
		edgeCoefs[2] = atp.e3;

		ifloat2 tileCoords[4];

		int chosenCoordTR[3];
		int chosenCoordTA[3];
		auto frameBufferWidth = csFrameWidth;
		auto frameBufferHeight =csFrameHeight;
		devGetAcceptRejectCoords(edgeCoefs, chosenCoordTR, chosenCoordTA);

		const float tileSize = 1.0f / CU_TILE_SIZE;
		for (int y = tileMiny; y <= tileMaxy; y++) {

			auto curTileY = y * frameBufferHeight / CU_TILE_SIZE;
			auto curTileY2 = (y + 1) * frameBufferHeight / CU_TILE_SIZE;
			auto cty1 = 1.0f * curTileY;
			auto cty2 = 1.0f * (curTileY2 - 1);

			for (int x = tileMinx; x <= tileMaxx; x++) {
				auto curTileX = x * frameBufferWidth / CU_TILE_SIZE;
				auto curTileX2 = (x + 1) * frameBufferWidth / CU_TILE_SIZE;
				auto ctx1 = 1.0f * curTileX;
				auto ctx2 = 1.0f * (curTileX2-1);

				tileCoords[VLT] = { ctx1, cty1 };
				tileCoords[VLB] = { ctx1, cty2 };
				tileCoords[VRB] = { ctx2, cty2 };
				tileCoords[VRT] = { ctx2, cty1 };

				int criteriaTR = 0;
				int criteriaTA = 0;
				for (int i = 0; i < 3; i++) {
					float criteriaTRLocal = edgeCoefs[i].x * tileCoords[chosenCoordTR[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTR[i]].y;
					float criteriaTALocal = edgeCoefs[i].x * tileCoords[chosenCoordTA[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTA[i]].y;
					if (criteriaTRLocal < -edgeCoefs[i].z) criteriaTR += 1;
					if (criteriaTALocal < -edgeCoefs[i].z) criteriaTA += 1;
				}
				if (criteriaTR != 3) {
					continue;
				}
				auto tileId = y * CU_TILE_SIZE + x;
				auto proposalId = 0;
				if (criteriaTA == 3) {
					proposalId = atomicAdd(&dCoverQueueCount[tileId], 1);
					proposalId = CU_SINGLE_TIME_TRIANGLE - 1 - proposalId;
				}
				else {
					proposalId = atomicAdd(&dRasterQueueCount[tileId], 1);
				}
				dRasterQueue[tileId][proposalId] = primitiveId;
			}
		}
	}

	
	IFRIT_DEVICE inline void devInterpolateVaryings(
		int id,
		const VaryingStore* const* dVaryingBuffer,
		const int indices[3],
		const float barycentric[3],
		VaryingStore& dest
	) {
		const auto va = dVaryingBuffer[id];
		VaryingStore vd;
		vd.vf4 = { 0,0,0,0 };
		for (int j = 0; j < 3; j++) {
			auto vaf4 = va[indices[j]].vf4;
			vd.vf4.x += vaf4.x * barycentric[j];
			vd.vf4.y += vaf4.y * barycentric[j];
			vd.vf4.z += vaf4.z * barycentric[j];
			vd.vf4.w += vaf4.w * barycentric[j];
		}
		dest = vd;
	}

	IFRIT_DEVICE void devPixelProcessingShadingPass(
		const AssembledTriangleProposalCUDA& atp,
		FragmentShader* fragmentShader,
		float bary[3],
		ifloat4** IFRIT_RESTRICT_CUDA dColorBuffer,
		const int* IFRIT_RESTRICT_CUDA dIndexBuffer,
		const VaryingStore* const* IFRIT_RESTRICT_CUDA dVaryingBuffer,
		int vertexStride,
		int varyingCount,
		int pixelPos
	) {
		ifloat4 colorOutputSingle;
		VaryingStore interpolatedVaryings[CU_MAX_VARYINGS];
		float desiredBary[3];
		desiredBary[0] = bary[0] * atp.b1.x + bary[1] * atp.b2.x + bary[2] * atp.b3.x;
		desiredBary[1] = bary[0] * atp.b1.y + bary[1] * atp.b2.y + bary[2] * atp.b3.y;
		desiredBary[2] = bary[0] * atp.b1.z + bary[1] * atp.b2.z + bary[2] * atp.b3.z;
		auto addr = dIndexBuffer + atp.originalPrimitive * vertexStride;
		for (int k = 0; k < varyingCount; k++) {
			devInterpolateVaryings(k, dVaryingBuffer, addr, desiredBary, interpolatedVaryings[k]);
		}
		fragmentShader->execute(interpolatedVaryings, &colorOutputSingle);
		dColorBuffer[0][pixelPos] = colorOutputSingle;
	}


	IFRIT_DEVICE void devTilingRasterizationChildProcess(
		uint32_t tileIdX,
		uint32_t tileIdY,
		uint32_t invoId,
		uint32_t totalBound,
		const AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		const uint32_t* IFRIT_RESTRICT_CUDA dRasterQueue,
		TileBinProposalCUDA* IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;

		auto globalInvocation = invoId;
		if (globalInvocation > totalBound)return;

		const auto tileId = tileIdY * CU_TILE_SIZE + tileIdX;
		const auto frameWidth = csFrameWidth;
		const auto frameHeight =csFrameHeight;

		const auto primitiveSrcId = dRasterQueue[globalInvocation];

		const auto& atri = dAssembledTriangles[primitiveSrcId];

		ifloat3 edgeCoefs[3];
		edgeCoefs[0] = atri.e1;
		edgeCoefs[1] = atri.e2;
		edgeCoefs[2] = atri.e3;

		int chosenCoordTR[3];
		int chosenCoordTA[3];
		devGetAcceptRejectCoords(edgeCoefs, chosenCoordTR, chosenCoordTA);

		auto curTileX = tileIdX * frameWidth / CU_TILE_SIZE;
		auto curTileY = tileIdY * frameHeight / CU_TILE_SIZE;
		auto curTileX2 = (tileIdX + 1) * frameWidth / CU_TILE_SIZE;
		auto curTileY2 = (tileIdY + 1) * frameHeight / CU_TILE_SIZE;
		auto curTileWid = curTileX2 - curTileX;
		auto curTileHei = curTileY2 - curTileY;

		const float dEps = CU_EPS * frameHeight * frameWidth;

		// Decomp into Sub Blocks
		
		for (int i = CU_SUBTILE_SIZE * CU_SUBTILE_SIZE - 1 - threadIdx.y; i >= 0; i--) {
			int criteriaTR = 0;
			int criteriaTA = 0;

			auto subTileIX = i % CU_SUBTILE_SIZE;
			auto subTileIY = i / CU_SUBTILE_SIZE;
			auto subTileTX = (tileIdX * CU_SUBTILE_SIZE + subTileIX);
			auto subTileTY = (tileIdY * CU_SUBTILE_SIZE + subTileIY);

			const int wp = (CU_SUBTILE_SIZE * CU_TILE_SIZE);
			int subTilePixelX = curTileX + (curTileWid * subTileIX >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelY = curTileY + (curTileHei * subTileIY >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelX2 = curTileX + (curTileWid * (subTileIX + 1) >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelY2 = curTileY + (curTileHei * (subTileIY + 1) >> CU_SUBTILE_SIZE_LOG);

			float subTileMinX = 1.0f * subTilePixelX;
			float subTileMinY = 1.0f * subTilePixelY;
			float subTileMaxX = 1.0f * (subTilePixelX2 - 1);
			float subTileMaxY = 1.0f * (subTilePixelY2 - 1);


			ifloat2 tileCoords[4];
			tileCoords[VLT] = { subTileMinX, subTileMinY };
			tileCoords[VLB] = { subTileMinX, subTileMaxY };
			tileCoords[VRB] = { subTileMaxX, subTileMaxY };
			tileCoords[VRT] = { subTileMaxX, subTileMinY };

			const float cmpf[3] = { dEps - edgeCoefs[0].z,dEps - edgeCoefs[1].z,dEps - edgeCoefs[2].z };
			for (int k = 0; k < 3; k++) {
				float criteriaTRLocal = edgeCoefs[k].x * tileCoords[chosenCoordTR[k]].x + edgeCoefs[k].y * tileCoords[chosenCoordTR[k]].y;
				float criteriaTALocal = edgeCoefs[k].x * tileCoords[chosenCoordTA[k]].x + edgeCoefs[k].y * tileCoords[chosenCoordTA[k]].y;
				criteriaTR += criteriaTRLocal < cmpf[k];
				criteriaTA += criteriaTALocal < cmpf[k];
			}

			if (criteriaTR != 3) {
				continue;
			}
			if (criteriaTA == 3) {
				TileBinProposalCUDA nprop;
				nprop.tileEnd = { (short)(subTilePixelX2 - 1),(short)(subTilePixelY2 - 1) };
				nprop.tile = { (short)subTilePixelX,(short)subTilePixelY };
				nprop.primId = primitiveSrcId;
				auto proposalInsIdx = atomicAdd(dCoverQueueCount, 1);
				dCoverQueue[proposalInsIdx] = nprop;
			}
			else {
				//Into Pixel level
				int wid = subTilePixelX2 - subTilePixelX;
				int hei = subTilePixelY2 - subTilePixelY;
				int tot = wid * hei;
				IFRIT_ASSUME(tot > 0);
				for (int i2 = tot - 1; i2 >= 0; i2--) {
					int dx = subTilePixelX + (uint32_t)i2 % (uint32_t)wid;
					int dy = subTilePixelY + (uint32_t)i2 / (uint32_t)wid;
					int accept = 0;
					for (int i = 0; i < 3; i++) {
						float criteria = edgeCoefs[i].x * dx + edgeCoefs[i].y * dy;
						accept += criteria < cmpf[i];
					}
					if (accept == 3) {
						TileBinProposalCUDA nprop;
						nprop.tileEnd = { (short)dx,(short)dy };
						nprop.tile = { (short)dx,(short)dy };
						nprop.primId = primitiveSrcId;
						auto proposalInsIdx = atomicAdd(dCoverQueueCount, 1);
						dCoverQueue[proposalInsIdx] = nprop;
					}
				}
			}
		}
	}
 
	// Kernel Implementations

	IFRIT_KERNEL void vertexProcessingKernel(
		VertexShader* vertexShader,
		uint32_t vertexCount,
		char* dVertexBuffer,
		TypeDescriptorEnum* dVertexTypeDescriptor,
		VaryingStore** dVaryingBuffer,
		TypeDescriptorEnum* dVaryingTypeDescriptor,
		ifloat4* dPosBuffer,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto globalInvoIdx = blockIdx.x * blockDim.x + threadIdx.x;
		if (globalInvoIdx >= vertexCount) return;
		const auto numAttrs = deviceConstants->attributeCount;
		const auto numVaryings = deviceConstants->varyingCount;

		const void* vertexInputPtrs[CU_MAX_ATTRIBUTES];
		VaryingStore* varyingOutputPtrs[CU_MAX_VARYINGS];
		
		for (int i = 0; i < numAttrs; i++) {
			vertexInputPtrs[i] = globalInvoIdx * csTotalVertexOffsets + dVertexBuffer + csVertexOffsets[i];
		}
		for (int i = 0; i < numVaryings; i++) {
			varyingOutputPtrs[i] = dVaryingBuffer[i] + globalInvoIdx;
		}
		vertexShader->execute(vertexInputPtrs, &dPosBuffer[globalInvoIdx], varyingOutputPtrs);
	}

	IFRIT_KERNEL void primaryBinnerRasterizerKernel(
		irect2Df* IFRIT_RESTRICT_CUDA dTileBounds,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		TileRasterDeviceConstants* deviceConstants
	) {

	}

	IFRIT_KERNEL void geometryProcessingKernel(
		ifloat4* IFRIT_RESTRICT_CUDA dPosBuffer,
		int* IFRIT_RESTRICT_CUDA dIndexBuffer,
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		uint32_t startingIndexId,
		uint32_t indexCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto globalInvoIdx = blockIdx.x * blockDim.x + threadIdx.x;
		if(globalInvoIdx >= indexCount / CU_TRIANGLE_STRIDE) return;

		const auto indexStart = globalInvoIdx * CU_TRIANGLE_STRIDE + startingIndexId;
		ifloat4 v1 = dPosBuffer[dIndexBuffer[indexStart]];
		ifloat4 v2 = dPosBuffer[dIndexBuffer[indexStart + 1]];
		ifloat4 v3 = dPosBuffer[dIndexBuffer[indexStart + 2]];
		if (csCounterClosewiseCull) {
			ifloat4 temp = v1;
			v1 = v3;
			v3 = temp;
		}
		const auto primId = globalInvoIdx + startingIndexId / CU_TRIANGLE_STRIDE;
		if (!devTriangleCull(v1, v2, v3)) {
			return;
		}
		using Ifrit::Engine::Math::ShaderOps::CUDA::dot;
		using Ifrit::Engine::Math::ShaderOps::CUDA::sub;
		using Ifrit::Engine::Math::ShaderOps::CUDA::add;
		using Ifrit::Engine::Math::ShaderOps::CUDA::multiply;
		using Ifrit::Engine::Math::ShaderOps::CUDA::lerp;

		constexpr uint32_t clipIts = 7;
		const ifloat4 clipCriteria[clipIts] = {
			{0,0,0,CU_EPS},
			{1,0,0,0},
			{-1,0,0,0},
			{0,1,0,0},
			{0,-1,0,0},
			{0,0,1,0},
			{0,0,-1,0}
		};

		TileRasterClipVertexCUDA retd[9];
		int retdIndex[14];
		int retdTriCnt = 3;

#define retidx(x,y) retdIndex[(x)*7+(y)]
#define ret(x,y) retd[retdIndex[(x)*7+(y)]]
		uint32_t retCnt[2] = { 0,3 };
		retd[0] = { {1,0,0},v1 };
		retd[1] = { {0,1,0},v2 };
		retd[2] = { {0,0,1},v3 };
		retidx(1,0) = 0;
		retidx(1,1) = 1;
		retidx(1,2) = 2;
		int clipTimes = 0;
		for (int i = 0; i < clipIts; i++) {
			ifloat4 outNormal = { clipCriteria[i].x,clipCriteria[i].y,clipCriteria[i].z,-1 };
			ifloat4 refPoint = { clipCriteria[i].x,clipCriteria[i].y,clipCriteria[i].z,clipCriteria[i].w };
			const auto cIdx = i & 1, cRIdx = 1 - (i & 1);
			retCnt[cIdx] = 0;
			const auto psize = retCnt[cRIdx];
			auto pc = ret(cRIdx, 0);
			auto npc = dot(pc.pos, outNormal);
			for (int j = 0; j < psize; j++) {
				const auto& pn = ret(cRIdx, (j + 1) % psize);
				auto npn = dot(pn.pos, outNormal);

				if (npc * npn < 0) {
					pc = ret(cRIdx, (j + psize- 1) % psize);
					ifloat4 dir = sub(pn.pos, pc.pos);
					float numo = pc.pos.w - pc.pos.x * refPoint.x - pc.pos.y * refPoint.y - pc.pos.z * refPoint.z;
					float deno = dir.x * refPoint.x + dir.y * refPoint.y + dir.z * refPoint.z - dir.w;
					float t = numo / deno;
					ifloat4 intersection = add(pc.pos, multiply(dir, t));
					ifloat3 barycenter = lerp(pc.barycenter, pn.barycenter, t);

					TileRasterClipVertexCUDA newp;
					newp.barycenter = barycenter;
					newp.pos = intersection;
					retd[retdTriCnt++] = newp;
					retidx(cIdx, retCnt[cIdx]++) = retdTriCnt - 1;
				}
				if (npn < CU_EPS) {
					retidx(cIdx, retCnt[cIdx]++) = (j + 1) % psize;
				}
				npc = npn;
			}
			if (retCnt[cIdx] < 3) {
				return;
			}
		}
		const auto clipOdd = clipTimes & 1;
		for (int i = 0; i < retCnt[clipOdd]; i++) {
			ret(clipOdd, i).pos.w = 1 / ret(clipOdd, i).pos.w;
			ret(clipOdd, i).pos.x *= ret(clipOdd, i).pos.w;
			ret(clipOdd, i).pos.y *= ret(clipOdd, i).pos.w;
			ret(clipOdd, i).pos.z *= ret(clipOdd, i).pos.w;

			ret(clipOdd, i).pos.x = ret(clipOdd, i).pos.x * 0.5f + 0.5f;
			ret(clipOdd, i).pos.y = ret(clipOdd, i).pos.y * 0.5f + 0.5f;
		}
		// Atomic Insertions
		auto threadId = threadIdx.x;

		const auto frameHeight = csFrameHeight;
		const auto frameWidth = csFrameWidth;

		auto idxSrc = atomicAdd(dAssembledTriangleCount, retCnt[clipOdd] - 2);
		const auto invFrameHeight = 1.0f / frameHeight;
		const auto invFrameWidth = 1.0f / frameWidth;
		for (int i = 0; i < retCnt[clipOdd] - 2; i++) {
			auto curIdx = idxSrc + i;
			AssembledTriangleProposalCUDA atri;
			atri.b1 = ret(clipOdd, 0).barycenter;
			atri.b2 = ret(clipOdd, i + 1).barycenter;
			atri.b3 = ret(clipOdd, i + 2).barycenter;
			const auto dv2 = ret(clipOdd, i + 1).pos;
			const auto dv3 = ret(clipOdd, i + 2).pos;
			const auto dv1 = ret(clipOdd, 0).pos;
			atri.v1 = dv1.z;
			atri.v2 = dv2.z;
			atri.v3 = dv3.z;

			const float ar = 1.0f / devEdgeFunction(dv1, dv2, dv3);;
			const float sV2V1y = dv2.y - dv1.y;
			const float sV2V1x = dv1.x - dv2.x;
			const float sV3V2y = dv3.y - dv2.y;
			const float sV3V2x = dv2.x - dv3.x;
			const float sV1V3y = dv1.y - dv3.y;
			const float sV1V3x = dv3.x - dv1.x;

			atri.f3 = { (float)(sV2V1y * ar) * dv3.w * invFrameHeight, (float)(sV2V1x * ar) * dv3.w * invFrameWidth,(float)((-dv1.x * sV2V1y - dv1.y * sV2V1x) * ar) * dv3.w };
			atri.f1 = { (float)(sV3V2y * ar) * dv1.w * invFrameHeight, (float)(sV3V2x * ar) * dv1.w * invFrameWidth,(float)((-dv2.x * sV3V2y - dv2.y * sV3V2x) * ar) * dv1.w };
			atri.f2 = { (float)(sV1V3y * ar) * dv2.w * invFrameHeight, (float)(sV1V3x * ar) * dv2.w * invFrameWidth,(float)((-dv3.x * sV1V3y - dv3.y * sV1V3x) * ar) * dv2.w };

			ifloat3 edgeCoefs[3];
			atri.e1 = { (float)(sV2V1y)*frameHeight,  (float)(sV2V1x)*frameWidth ,  (float)(dv2.x * dv1.y - dv1.x * dv2.y) * frameHeight * frameWidth };
			atri.e2 = { (float)(sV3V2y)*frameHeight,  (float)(sV3V2x)*frameWidth ,  (float)(dv3.x * dv2.y - dv2.x * dv3.y) * frameHeight * frameWidth };
			atri.e3 = { (float)(sV1V3y)*frameHeight,  (float)(sV1V3x)*frameWidth ,  (float)(dv1.x * dv3.y - dv3.x * dv1.y) * frameHeight * frameWidth };

			atri.originalPrimitive = primId;
			irect2Df bbox;
			if (!devTriangleSimpleClip(dv1, dv2, dv3, bbox)) continue;
			if constexpr (CU_NOT_OPT_TILED_BINNER) {
				devExecuteBinner(idxSrc + i, atri, bbox, dRasterQueue, dRasterQueueCount, dCoverQueue, dCoverQueueCount, deviceConstants);
			}
			dAssembledTriangles[curIdx] = atri;
		}
#undef ret
#undef retidx

	}

	IFRIT_KERNEL void secondaryBinnerRasterizerKernel(
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto tileIdxX = blockIdx.x ;
		const auto tileIdxY = blockIdx.y;
		const auto threadX = threadIdx.x;
		const auto blockX = blockDim.x;
		const auto tileId = tileIdxY * CU_TILE_SIZE+ tileIdxX;
		IFRIT_SHARED uint32_t sdAtomicCounter[1];
		const auto oldCounter = dCoverQueueCount[tileId];
		if (threadX == 0) {
			sdAtomicCounter[0] = 0;
		}
		__syncthreads();
		const auto dRaster = dRasterQueue[tileId];
		const auto dCover = dCoverQueue[tileId];
		const auto sdRastCandidates = dRasterQueueCount[tileId];

		for (int i = threadX; i < sdRastCandidates; i+= blockX) {
			devTilingRasterizationChildProcess(tileIdxX, tileIdxY, i, sdRastCandidates, dAssembledTriangles,
				dRaster, dCover, sdAtomicCounter, deviceConstants);
			
		}
		__syncthreads();
		if (threadX == 0) {
			dCoverQueueCount[tileId] = sdAtomicCounter[0];
			dRasterQueueCount[tileId] = oldCounter;
		}
	}


	IFRIT_KERNEL void fragmentShadingKernelPerTile(
		FragmentShader*  fragmentShader,
		int* IFRIT_RESTRICT_CUDA dIndexBuffer,
		const VaryingStore* const* IFRIT_RESTRICT_CUDA dVaryingBuffer,
		const TileBinProposalCUDA* const* IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembleTriangleCounter,
		const AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAtp,
		ifloat4** IFRIT_RESTRICT_CUDA dColorBuffer,
		float* IFRIT_RESTRICT_CUDA dDepthBuffer,
		const TileRasterDeviceConstants* deviceConstants
	) {
		uint32_t tileX = blockIdx.x;
		uint32_t tileY = blockIdx.y;

		uint32_t tileId = tileY * CU_TILE_SIZE + tileX;
		const auto frameWidth = csFrameWidth;
		const auto frameHeight = csFrameHeight;
		const auto candidates = dCoverQueueCount[tileId];
		const auto completeCandidates = dRasterQueueCount[tileId];
		constexpr auto vertexStride = CU_TRIANGLE_STRIDE;
		const auto varyingCount = deviceConstants->varyingCount;

		const int threadX = threadIdx.x;
		const int threadY = threadIdx.y;
		const int blockX = blockDim.x;
		const int blockY = blockDim.y;
		const int bds = blockDim.x * blockDim.y;
		const auto threadId = threadY * bds + threadX;

		const int pixelXS = threadX + tileX * csFrameWidth / CU_TILE_SIZE;
		const int pixelYS = threadY + tileY * csFrameHeight / CU_TILE_SIZE;
		
		const TileBinProposalCUDA* sdCoverQueueSrc = dCoverQueue[tileId];
		const uint32_t* sdRasterQueueSrc = dRasterQueue[tileId];

		float localDepthBuffer = 1;
		float candidateBary[3];
		int candidatePrim = -1;
		const float compareDepth = dDepthBuffer[pixelYS * frameWidth + pixelXS];
		float pDx = 1.0f * pixelXS;
		float pDy = 1.0f * pixelYS;

		for (int i = completeCandidates - 1; i >= 0; i--) {
			const auto proposal = sdRasterQueueSrc[CU_SINGLE_TIME_TRIANGLE - 1 - i];
			const auto atp = dAtp[proposal];
			float pos[4];
			pos[0] = atp.v1;
			pos[1] = atp.v2;
			pos[2] = atp.v3;

			float bary[3];
			float interpolatedDepth;

			bary[0] = (atp.f1.x * pDx + atp.f1.y * pDy + atp.f1.z);
			bary[1] = (atp.f2.x * pDx + atp.f2.y * pDy + atp.f2.z);
			bary[2] = (atp.f3.x * pDx + atp.f3.y * pDy + atp.f3.z);
			interpolatedDepth = bary[0] * pos[0] + bary[1] * pos[1] + bary[2] * pos[2];
			float zCorr = 1.0f / (bary[0] + bary[1] + bary[2]);
			interpolatedDepth *= zCorr;
			if (interpolatedDepth <= localDepthBuffer) {
				localDepthBuffer = interpolatedDepth;
				candidatePrim = proposal;
				candidateBary[0] = bary[0] * zCorr;
				candidateBary[1] = bary[1] * zCorr;
				candidateBary[2] = bary[2] * zCorr;
			}
		}
		for (int i = candidates - 1; i >= 0; i--) {
			const auto proposal = sdCoverQueueSrc[i];
			const auto atp = dAtp[proposal.primId];
			const auto startX = proposal.tile.x;
			const auto startY = proposal.tile.y;
			const auto endX = proposal.tileEnd.x;
			const auto endY = proposal.tileEnd.y;

			if (startX <= pixelXS && pixelXS <= endX && startY <= pixelYS && pixelYS <= endY) {
				// Z PrePas
				float pos[4];
				pos[0] = atp.v1;
				pos[1] = atp.v2;
				pos[2] = atp.v3;

				float bary[3];
				float interpolatedDepth;

				bary[0] = (atp.f1.x * pDx + atp.f1.y * pDy + atp.f1.z);
				bary[1] = (atp.f2.x * pDx + atp.f2.y * pDy + atp.f2.z);
				bary[2] = (atp.f3.x * pDx + atp.f3.y * pDy + atp.f3.z);
				interpolatedDepth = bary[0] * pos[0] + bary[1] * pos[1] + bary[2] * pos[2];
				float zCorr = 1.0f / (bary[0] + bary[1] + bary[2]);
				interpolatedDepth *= zCorr;
				if (interpolatedDepth <= localDepthBuffer) {
					localDepthBuffer = interpolatedDepth;
					candidatePrim = proposal.primId;
					candidateBary[0] = bary[0] * zCorr;
					candidateBary[1] = bary[1] * zCorr;
					candidateBary[2] = bary[2] * zCorr;
				}
			}
		}
		if (candidatePrim != -1 && localDepthBuffer< compareDepth) {
			devPixelProcessingShadingPass(dAtp[candidatePrim], fragmentShader, candidateBary, dColorBuffer, dIndexBuffer,
				dVaryingBuffer, vertexStride, varyingCount, pixelYS * frameWidth + pixelXS);
			dDepthBuffer[pixelYS * frameWidth + pixelXS] = localDepthBuffer;
		}

		//Reset kernels
		if (threadX == 0) {
			dCoverQueueCount[tileId] = 0;
			dRasterQueueCount[tileId] = 0;
			dAssembleTriangleCounter[0] = 0;
		}

	}


	IFRIT_KERNEL void imageResetFloat32Kernel(
		float* dBuffer,
		uint32_t channels,
		float value
	) {
		const auto invoX = blockIdx.x * blockDim.x + threadIdx.x;
		const auto invoY = blockIdx.y * blockDim.y + threadIdx.y;
		if (invoX >= csFrameWidth || invoY >= csFrameHeight) {
			return;
		}
		for(int i=0;i<channels;i++) {
			dBuffer[(invoY * csFrameWidth + invoX) * channels + i] = value;
		}
	}

	IFRIT_KERNEL void imageResetInt32Kernel(
		int* dBuffer,
		uint32_t imageX,
		uint32_t imageY,
		uint32_t channels,
		int value
	) {
		const auto invoX = blockIdx.x * blockDim.x + threadIdx.x;
		const auto invoY = blockIdx.y * blockDim.y + threadIdx.y;
		if (invoX >= imageX || invoY >= imageY) {
			return;
		}
		for (int i = 0; i < channels; i++) {
			dBuffer[(invoY * imageX + invoX) * channels + i] = value;
		}
	}

	IFRIT_KERNEL void resetKernel(
		uint32_t* count,
		uint32_t size
	) {
		const auto globalInvocation = blockIdx.x * blockDim.x + threadIdx.x;
		if (globalInvocation >= size) {
			return;
		}
		count[globalInvocation] = 0;
	}

	IFRIT_KERNEL void testingKernel() {
		printf("Hello World\n");
	}
}


namespace  Ifrit::Engine::TileRaster::CUDA::Invocation {


	template<typename T>
	__global__ void kernFixVTable(T* devicePtr) {
		T temp(*devicePtr);
		memcpy(devicePtr, &temp, sizeof(T));
	}

	template<typename T>
	__host__ T* hostGetDeviceObjectCopy(T* hostObject) {
		T* deviceHandle;
		hipMalloc(&deviceHandle, sizeof(T));
		hipMemcpy(deviceHandle, hostObject, sizeof(T), hipMemcpyHostToDevice);
		printf("Copying object to CUDA, %lld,%d\n", deviceHandle, 1);
		kernFixVTable<T> CU_KARG2(1, 1)(deviceHandle);
		hipDeviceSynchronize();
		printf("Cuda Addr=%lld\n", deviceHandle);
		printf("Object copied to CUDA\n");
		return deviceHandle;
	}

	template<class T>
	T* copyShaderToDevice(T* x) {
		return hostGetDeviceObjectCopy<T>(x);
	}

	void testingKernelWrapper() {
		Impl::testingKernel CU_KARG2(4,4) ();
		hipDeviceSynchronize();
	}
	char* deviceMalloc(uint32_t size) {
		char* ptr;
		hipMalloc(&ptr, size);
		return ptr;
	
	}
	void deviceFree(char* ptr) {
		hipFree(ptr);
	}


	int* getIndexBufferDeviceAddr(const int* hIndexBuffer, uint32_t indexBufferSize, int* dOldIndexBuffer) {
		if(dOldIndexBuffer != nullptr) {
			hipFree(dOldIndexBuffer);
		}
		int* dIndexBuffer;
		hipMalloc(&dIndexBuffer, indexBufferSize * sizeof(int));
		hipMemcpy(dIndexBuffer, hIndexBuffer, indexBufferSize * sizeof(int), hipMemcpyHostToDevice);
		return dIndexBuffer;
	}
	char* getVertexBufferDeviceAddr(const char* hVertexBuffer, uint32_t bufferSize, char* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		char* dBuffer;
		hipMalloc(&dBuffer, bufferSize);
		hipMemcpy(dBuffer, hVertexBuffer, bufferSize, hipMemcpyHostToDevice);
		return dBuffer;
	
	}
	TypeDescriptorEnum* getTypeDescriptorDeviceAddr(const TypeDescriptorEnum* hBuffer, uint32_t bufferSize, TypeDescriptorEnum* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		TypeDescriptorEnum* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(TypeDescriptorEnum));
		hipMemcpy(dBuffer, hBuffer, bufferSize * sizeof(TypeDescriptorEnum), hipMemcpyHostToDevice);
		return dBuffer;
	
	}
	float* getDepthBufferDeviceAddr(uint32_t bufferSize, float* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		float* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(float));
		return dBuffer;
	}
	ifloat4* getPositionBufferDeviceAddr(uint32_t bufferSize, ifloat4* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		ifloat4* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(ifloat4));
		return dBuffer;
	
	}
	int* getShadingLockDeviceAddr(uint32_t bufferSize, int* dOldBuffer) {
		if (dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		int* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(int));
		return dBuffer;
	}
	void getColorBufferDeviceAddr(
		const std::vector<ifloat4*>& hColorBuffer,
		std::vector<ifloat4*>& dhColorBuffer,
		ifloat4**& dColorBuffer,
		uint32_t bufferSize,
		std::vector<ifloat4*>& dhOldColorBuffer,
		ifloat4** dOldBuffer
	) {
		//TODO: POTENTIAL BUGS & CUDA DEVICE MEMORY LEAK
		if (dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		dhColorBuffer.resize(hColorBuffer.size());

		hipMalloc(&dColorBuffer, hColorBuffer.size() * sizeof(ifloat4*));
		for (int i = 0; i < hColorBuffer.size(); i++) {
			hipMalloc(&dhColorBuffer[i], bufferSize * sizeof(ifloat4));
			hipMemcpy(dhColorBuffer[i], hColorBuffer[i], bufferSize * sizeof(ifloat4), hipMemcpyHostToDevice);
		}
		hipMemcpy(dColorBuffer, dhColorBuffer.data(), hColorBuffer.size() * sizeof(ifloat4*), hipMemcpyHostToDevice);
	

	}
	void updateFrameBufferConstants(uint32_t width,uint32_t height) {
		hipMemcpyToSymbol(HIP_SYMBOL(Impl::csFrameWidth), &width, sizeof(uint32_t));
		hipMemcpyToSymbol(HIP_SYMBOL(Impl::csFrameHeight), &height, sizeof(uint32_t));
		Impl::hsFrameHeight = height;
		Impl::hsFrameWidth = width;
	}

	void initCudaRendering() {
		hipMemcpyToSymbol(HIP_SYMBOL(Impl::csCounterClosewiseCull), &Impl::hsCounterClosewiseCull, sizeof(Impl::hsCounterClosewiseCull));
	}

	void updateVertexLayout(TypeDescriptorEnum* dVertexTypeDescriptor, int attrCounts) {
		Impl::hsTotalVertexOffsets = 0;
		for (int i = 0; i < attrCounts; i++) {
			int cof = 0;
			Impl::hsVertexOffsets[i] = Impl::hsTotalVertexOffsets;
			if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT1) cof = sizeof(float);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT2) cof = sizeof(ifloat2);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT3) cof = sizeof(ifloat3);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT4)cof = sizeof(ifloat4);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT1) cof = sizeof(int);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT2) cof = sizeof(iint2);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT3) cof = sizeof(iint3);
			else if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT4) cof = sizeof(iint4);
			Impl::hsTotalVertexOffsets += cof;
		}
		hipMemcpyToSymbol(HIP_SYMBOL(Impl::csVertexOffsets), Impl::hsVertexOffsets, sizeof(Impl::hsVertexOffsets));
		hipMemcpyToSymbol(HIP_SYMBOL(Impl::csTotalVertexOffsets), &Impl::hsTotalVertexOffsets, sizeof(Impl::hsTotalVertexOffsets));
	}

	void invokeCudaRendering(
		char* dVertexBuffer,
		TypeDescriptorEnum* dVertexTypeDescriptor,
		TypeDescriptorEnum* dVaryingTypeDescriptor,
		int* dIndexBuffer,
		int* dShaderLockBuffer,
		VertexShader* dVertexShader,
		FragmentShader* dFragmentShader,
		ifloat4** dColorBuffer,
		ifloat4** dHostColorBuffer,
		ifloat4** hColorBuffer,
		uint32_t dHostColorBufferSize,
		float* dDepthBuffer,
		ifloat4* dPositionBuffer,
		TileRasterDeviceConstants* deviceConstants,
		TileRasterDeviceContext* deviceContext,
		bool doubleBuffering,
		ifloat4** dLastColorBuffer
	) IFRIT_AP_NOTHROW {
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		hipMemcpy(deviceContext->dDeviceConstants, deviceConstants, sizeof(TileRasterDeviceConstants), hipMemcpyHostToDevice);

		// Stream Preparation
		static int initFlag = 0;
		static hipStream_t copyStream, computeStream;
		static hipEvent_t  copyStart, copyEnd;
		if (initFlag == 0) {
			initFlag = 1;
			hipStreamCreate(&copyStream);
			hipStreamCreate(&computeStream);
			hipEventCreate(&copyStart);
			hipEventCreate(&copyEnd);
		}
		
		// Compute
		std::chrono::steady_clock::time_point end1 = std::chrono::steady_clock::now();
		const int tileSizeX = (Impl::hsFrameWidth / CU_TILE_SIZE) + ((Impl::hsFrameWidth % CU_TILE_SIZE) != 0);
		const int tileSizeY = (Impl::hsFrameHeight / CU_TILE_SIZE) + ((Impl::hsFrameHeight % CU_TILE_SIZE) != 0);

		constexpr int dispatchThreadsX = 8;
		constexpr int dispatchThreadsY = 8;
		int dispatchBlocksX = (Impl::hsFrameWidth / dispatchThreadsX) + ((Impl::hsFrameWidth % dispatchThreadsX) != 0);
		int dispatchBlocksY = (Impl::hsFrameHeight / dispatchThreadsY) + ((Impl::hsFrameHeight % dispatchThreadsY) != 0);

		Impl::imageResetFloat32Kernel CU_KARG4(dim3(dispatchBlocksX, dispatchBlocksY), dim3(dispatchThreadsX, dispatchThreadsY), 0, computeStream)(
			dDepthBuffer, 1, 255.0f
		);
		for (int i = 0; i < dHostColorBufferSize; i++) {
			Impl::imageResetFloat32Kernel CU_KARG4(dim3(dispatchBlocksX, dispatchBlocksY), dim3(dispatchThreadsX, dispatchThreadsY), 0, computeStream)(
				(float*)dHostColorBuffer[i], 4, 0.0f
			);
		}
		int vertexExecutionBlocks = (deviceConstants->vertexCount / CU_VERTEX_PROCESSING_THREADS) + ((deviceConstants->vertexCount % CU_VERTEX_PROCESSING_THREADS) != 0);
		Impl::vertexProcessingKernel CU_KARG4(vertexExecutionBlocks, CU_VERTEX_PROCESSING_THREADS, 0, computeStream)(
			dVertexShader, deviceConstants->vertexCount, dVertexBuffer, dVertexTypeDescriptor,
			deviceContext->dVaryingBuffer, dVaryingTypeDescriptor, dPositionBuffer, deviceContext->dDeviceConstants
		);
		
		constexpr int totalTiles = CU_TILE_SIZE * CU_TILE_SIZE;
		Impl::resetKernel CU_KARG4(1, 1, 0, computeStream)(deviceContext->dAssembledTrianglesCounter2, totalTiles);
		Impl::resetKernel CU_KARG4(CU_TILE_SIZE, CU_TILE_SIZE, 0, computeStream)(deviceContext->dRasterQueueCounter, totalTiles);
		Impl::resetKernel CU_KARG4(CU_TILE_SIZE, CU_TILE_SIZE, 0, computeStream)(deviceContext->dCoverQueueCounter, totalTiles);
		for (int i = 0; i < deviceConstants->totalIndexCount; i += CU_SINGLE_TIME_TRIANGLE * 3) {
			auto indexCount = std::min(CU_SINGLE_TIME_TRIANGLE * 3, deviceConstants->totalIndexCount - i);
			int geometryExecutionBlocks = (indexCount / CU_TRIANGLE_STRIDE / CU_GEOMETRY_PROCESSING_THREADS) + ((indexCount / CU_TRIANGLE_STRIDE % CU_GEOMETRY_PROCESSING_THREADS) != 0);
			Impl::geometryProcessingKernel CU_KARG4(geometryExecutionBlocks, CU_GEOMETRY_PROCESSING_THREADS, 0, computeStream)(
				dPositionBuffer, dIndexBuffer, deviceContext->dAssembledTriangles2, deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dRasterQueue,deviceContext->dRasterQueueCounter, deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter,i, indexCount,
				deviceContext->dDeviceConstants
			);

			Impl::secondaryBinnerRasterizerKernel CU_KARG4(dim3(CU_TILE_SIZE, CU_TILE_SIZE, 1), dim3(CU_RASTERIZATION_THREADS_PER_TILE, 1, 1), 0, computeStream)(
				deviceContext->dAssembledTriangles2, deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dRasterQueue, deviceContext->dRasterQueueCounter, deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter, 
				deviceContext->dDeviceConstants
			);

			Impl::fragmentShadingKernelPerTile CU_KARG4(dim3(CU_TILE_SIZE, CU_TILE_SIZE, 1), dim3(tileSizeX, tileSizeY, 1), 0, computeStream) (
				dFragmentShader, dIndexBuffer, deviceContext->dVaryingBuffer,
				deviceContext->dCoverQueue2, deviceContext->dRasterQueue, deviceContext->dCoverQueueCounter, deviceContext->dRasterQueueCounter, 
				deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dAssembledTriangles2, dColorBuffer, dDepthBuffer, deviceContext->dDeviceConstants
			);

		}
		if (!doubleBuffering) {
			hipDeviceSynchronize();
		}

		// Memory Copy
		std::chrono::steady_clock::time_point end2 = std::chrono::steady_clock::now();
		if (doubleBuffering) {
			for (int i = 0; i < dHostColorBufferSize; i++) {
				hipMemcpyAsync(hColorBuffer[i], dLastColorBuffer[i], Impl::hsFrameWidth * Impl::hsFrameHeight * sizeof(ifloat4), hipMemcpyDeviceToHost, copyStream);
			}
		}
		hipStreamSynchronize(computeStream);
		if (doubleBuffering) {
			hipStreamSynchronize(copyStream);
		}

		if (!doubleBuffering) {
			for (int i = 0; i < dHostColorBufferSize; i++) {
				hipMemcpy(hColorBuffer[i], dHostColorBuffer[i], Impl::csFrameWidth * Impl::csFrameHeight * sizeof(ifloat4), hipMemcpyDeviceToHost);
			}
		}
		std::chrono::steady_clock::time_point end3 = std::chrono::steady_clock::now();

		// End of rendering
		auto memcpyTimes = std::chrono::duration_cast<std::chrono::microseconds>(end1 - begin).count();
		auto computeTimes = std::chrono::duration_cast<std::chrono::microseconds>(end2 - end1).count();
		auto copybackTimes = std::chrono::duration_cast<std::chrono::microseconds>(end3 - end2).count();

		static long long w = 0;
		static long long wt = 0;
		w += copybackTimes;
		wt += 1;
		printf("AvgTime:%lld\n", w / wt);
		//printf("Memcpy,Compute,Copyback,Counter: %lld,%lld,%lld,%d\n", memcpyTimes, computeTimes, copybackTimes,cntw);
	}
}