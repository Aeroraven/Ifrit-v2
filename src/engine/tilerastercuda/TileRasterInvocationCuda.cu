#include "hip/hip_runtime.h"
#include "engine/tilerastercuda/TileRasterInvocationCuda.cuh"
#include "engine/math/ShaderOpsCuda.cuh"
#include "engine/tilerastercuda/TileRasterDeviceContextCuda.cuh"
#include "engine/tilerastercuda/TileRasterConstantsCuda.h"
namespace Ifrit::Engine::TileRaster::CUDA::Invocation::Impl {
	struct FragmentShadingPos {
		int tileId;
		int candidateId;
	};
	struct FragmentShadingQueue {
		int curTile = 0;
		int curCandidate = 0;
		int lock = 0;
	};

	IFRIT_DEVICE float devEdgeFunction(ifloat4 a, ifloat4 b, ifloat4 c) {
		return (c.x - a.x) * (b.y - a.y) - (c.y - a.y) * (b.x - a.x);
	}
	IFRIT_DEVICE bool devTriangleCull(ifloat4 v1, ifloat4 v2, ifloat4 v3) {
		float d1 = (v1.x * v2.y);
		float d2 = (v2.x * v3.y);
		float d3 = (v3.x * v1.y);
		float n1 = (v3.x * v2.y);
		float n2 = (v1.x * v3.y);
		float n3 = (v2.x * v1.y);
		float d = d1 + d2 + d3 - n1 - n2 - n3;
		if (d < 0.0f) return false;
		return true;
	}

	IFRIT_DEVICE void devGetAcceptRejectCoords(ifloat3 edgeCoefs[3], int chosenCoordTR[3], int chosenCoordTA[3]) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;
		for (int i = 0; i < 3; i++) {
			bool normalRight = edgeCoefs[i].x < 0;
			bool normalDown = edgeCoefs[i].y < 0;
			if (normalRight) {
				if (normalDown) {
					chosenCoordTR[i] = VRB;
					chosenCoordTA[i] = VLT;
				}
				else {
					chosenCoordTR[i] = VRT;
					chosenCoordTA[i] = VLB;
				}
			}
			else {
				if (normalDown) {
					chosenCoordTR[i] = VLB;
					chosenCoordTA[i] = VRT;
				}
				else {
					chosenCoordTR[i] = VLT;
					chosenCoordTA[i] = VRB;
				}
			}
		}
	}

	IFRIT_DEVICE bool devTriangleSimpleClip(ifloat4 v1, ifloat4 v2, ifloat4 v3, irect2Df& bbox) {
		bool inside = true;
		float minx = min(v1.x, min(v2.x, v3.x));
		float miny = min(v1.y, min(v2.y, v3.y));
		float maxx = max(v1.x, max(v2.x, v3.x));
		float maxy = max(v1.y, max(v2.y, v3.y));
		float maxz = max(v1.z, max(v2.z, v3.z));
		float minz = min(v1.z, min(v2.z, v3.z));
		if (maxz < 0.0f) return false;
		if (minz > 1.0f) return false;
		if (maxx < -1.0f) return false;
		if (minx > 1.0f) return false;
		if (maxy < -1.0f) return false;
		if (miny > 1.0f) return false;
		bbox.x = minx;
		bbox.y = miny;
		bbox.w = maxx - minx;
		bbox.h = maxy - miny;
		return true;
	}
	IFRIT_DEVICE void devExecuteBinner(
		int primitiveId,
		AssembledTriangleProposalCUDA& atp,
		irect2Df bbox,
		uint32_t** dRasterQueue,
		uint32_t* dRasterQueueCount,
		TileBinProposalCUDA** dCoverQueue,
		uint32_t* dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;
		float minx = bbox.x ;
		float miny = bbox.y ;
		float maxx = (bbox.x + bbox.w);
		float maxy = (bbox.y + bbox.h);

		int tileMinx = max(0, (int)(minx * CU_TILE_SIZE));
		int tileMiny = max(0, (int)(miny * CU_TILE_SIZE));
		int tileMaxx = min(CU_TILE_SIZE - 1, (int)(maxx * CU_TILE_SIZE));
		int tileMaxy = min(CU_TILE_SIZE - 1, (int)(maxy * CU_TILE_SIZE));

		ifloat3 edgeCoefs[3];
		edgeCoefs[0] = atp.e1;
		edgeCoefs[1] = atp.e2;
		edgeCoefs[2] = atp.e3;

		ifloat2 tileCoords[4];

		int chosenCoordTR[3];
		int chosenCoordTA[3];
		auto frameBufferWidth = deviceConstants->frameBufferWidth;
		auto frameBufferHeight = deviceConstants->frameBufferHeight;
		devGetAcceptRejectCoords(edgeCoefs, chosenCoordTR, chosenCoordTA);

		const float tileSize = 1.0f / CU_TILE_SIZE;
		for (int y = tileMiny; y <= tileMaxy; y++) {

			auto curTileY = y * frameBufferHeight / CU_TILE_SIZE;
			auto curTileY2 = (y + 1) * frameBufferHeight / CU_TILE_SIZE;
			auto cty1 = 1.0f * curTileY;
			auto cty2 = 1.0f * (curTileY2 - 1);

			for (int x = tileMinx; x <= tileMaxx; x++) {
				auto curTileX = x * frameBufferWidth / CU_TILE_SIZE;
				auto curTileX2 = (x + 1) * frameBufferWidth / CU_TILE_SIZE;
				auto ctx1 = 1.0f * curTileX;
				auto ctx2 = 1.0f * (curTileX2-1);

				tileCoords[VLT] = { ctx1, cty1 };
				tileCoords[VLB] = { ctx1, cty2 };
				tileCoords[VRB] = { ctx2, cty2 };
				tileCoords[VRT] = { ctx2, cty1 };

				int criteriaTR = 0;
				int criteriaTA = 0;
				for (int i = 0; i < 3; i++) {
					float criteriaTRLocal = edgeCoefs[i].x * tileCoords[chosenCoordTR[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTR[i]].y;
					float criteriaTALocal = edgeCoefs[i].x * tileCoords[chosenCoordTA[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTA[i]].y;
					if (criteriaTRLocal < -edgeCoefs[i].z) criteriaTR += 1;
					if (criteriaTALocal < -edgeCoefs[i].z) criteriaTA += 1;
				}
				if (criteriaTR != 3) {
					continue;
				}
				auto workerId = threadIdx.x;
				auto tileId = y * CU_TILE_SIZE + x;
				if (criteriaTA == 3) {
					TileBinProposalCUDA proposal;
					proposal.tileEnd = { (short)(curTileX2-1),(short)(curTileY2-1) };
					proposal.primId = primitiveId;
					proposal.tile = { (short)curTileX,(short)curTileY };
					auto tileId = y * CU_TILE_SIZE + x;
					auto proposalId = atomicAdd(&dCoverQueueCount[tileId], 1);
					dCoverQueue[tileId][proposalId] = proposal;
				}
				else {
					auto proposalId = atomicAdd(&dRasterQueueCount[tileId], 1);
					dRasterQueue[tileId][proposalId] = primitiveId;
				}
			}
		}
	}


	IFRIT_DEVICE int devTriangleHomogeneousClip(
		const int primitiveId,
		ifloat4 v1,
		ifloat4 v2,
		ifloat4 v3,
		AssembledTriangleProposalCUDA* dProposals,
		uint32_t* dProposalCount,
		float frameWidth,
		float frameHeight,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCounts,
		TileRasterDeviceConstants* deviceConstants
	) {
		using Ifrit::Engine::Math::ShaderOps::CUDA::dot;
		using Ifrit::Engine::Math::ShaderOps::CUDA::sub;
		using Ifrit::Engine::Math::ShaderOps::CUDA::add;
		using Ifrit::Engine::Math::ShaderOps::CUDA::multiply;
		using Ifrit::Engine::Math::ShaderOps::CUDA::lerp;

		constexpr uint32_t clipIts = 7;
		const ifloat4 clipCriteria[clipIts] = {
			{0,0,0,CU_EPS},
			{1,0,0,0},
			{-1,0,0,0},
			{0,1,0,0},
			{0,-1,0,0},
			{0,0,1,0},
			{0,0,-1,0}
		};
		TileRasterClipVertexCUDA retd[18];
#define ret(x,y) retd[(x)*9+(y)]
		uint32_t retCnt[2] = { 0,3 };
		ret(1,0) = {{1,0,0},v1};
		ret(1,1) = {{0,1,0},v2};
		ret(1,2) = {{0,0,1},v3};
		int clipTimes = 0;
		for (int i = 0; i < clipIts; i++) {
			ifloat4 outNormal = { clipCriteria[i].x,clipCriteria[i].y,clipCriteria[i].z,-1 };
			ifloat4 refPoint = { clipCriteria[i].x,clipCriteria[i].y,clipCriteria[i].z,clipCriteria[i].w };
			const auto cIdx = i & 1, cRIdx = 1 - (i & 1);
			retCnt[cIdx] = 0;
			const auto psize = retCnt[cRIdx];
			if (psize == 0) {
				return 0;
			}
			auto pc = ret(cRIdx,0);
			auto npc = dot(pc.pos, outNormal);
			for (int j = 0; j < psize; j++) {
				const auto& pn = ret(cRIdx,(j + 1) % psize);
				auto npn = dot(pn.pos, outNormal);

				if (npc * npn < 0) {
					ifloat4 dir = sub(pn.pos, pc.pos);
					float numo = pc.pos.w - pc.pos.x * refPoint.x - pc.pos.y * refPoint.y - pc.pos.z * refPoint.z;
					float deno = dir.x * refPoint.x + dir.y * refPoint.y + dir.z * refPoint.z - dir.w;
					float t = numo / deno;
					ifloat4 intersection = add(pc.pos, multiply(dir, t));
					ifloat3 barycenter = lerp(pc.barycenter, pn.barycenter, t);

					TileRasterClipVertexCUDA newp;
					newp.barycenter = barycenter;
					newp.pos = intersection;
					ret(cIdx,retCnt[cIdx]++) = (newp);
				}
				if (npn < CU_EPS) {
					ret(cIdx,retCnt[cIdx]++) = pn;
				}
				pc = pn;
				npc = npn;
			}
			if (retCnt[cIdx] < 3) {
				return 0;
			}
		}
		const auto clipOdd = clipTimes & 1;
		for (int i = 0; i < retCnt[clipOdd]; i++) {
			ret(clipOdd,i).pos.w = 1 / ret(clipOdd,i).pos.w;
			ret(clipOdd,i).pos.x *= ret(clipOdd,i).pos.w;
			ret(clipOdd,i).pos.y *= ret(clipOdd,i).pos.w;
			ret(clipOdd,i).pos.z *= ret(clipOdd,i).pos.w;


			ret(clipOdd,i).pos.x = ret(clipOdd,i).pos.x * 0.5 + 0.5;
			ret(clipOdd,i).pos.y = ret(clipOdd,i).pos.y * 0.5 + 0.5;
		}
		// Atomic Insertions
		auto threadId = threadIdx.x;

		auto idxSrc = atomicAdd(dProposalCount, retCnt[clipOdd] - 2);
		for (int i = 0; i < retCnt[clipOdd] - 2; i++) {
			auto curIdx = idxSrc + i;
			AssembledTriangleProposalCUDA atri;
			atri.b1 = ret(clipOdd, 0).barycenter;
			atri.b2 = ret(clipOdd, i + 1).barycenter;
			atri.b3 = ret(clipOdd, i + 2).barycenter;
			atri.v1 = ret(clipOdd, 0).pos;
			atri.v2 = ret(clipOdd, i + 1).pos;
			atri.v3 = ret(clipOdd, i + 2).pos;

			const float ar = 1.0f / devEdgeFunction(atri.v1, atri.v2, atri.v3);
			const float sV2V1y = atri.v2.y - atri.v1.y;
			const float sV2V1x = atri.v1.x - atri.v2.x;
			const float sV3V2y = atri.v3.y - atri.v2.y;
			const float sV3V2x = atri.v2.x - atri.v3.x;
			const float sV1V3y = atri.v1.y - atri.v3.y;
			const float sV1V3x = atri.v3.x - atri.v1.x;

			atri.f3 = { (float)(sV2V1y * ar) * atri.v3.w / frameHeight, (float)(sV2V1x * ar) * atri.v3.w / frameWidth,(float)((-atri.v1.x * sV2V1y - atri.v1.y * sV2V1x) * ar) * atri.v3.w };
			atri.f1 = { (float)(sV3V2y * ar) * atri.v1.w / frameHeight, (float)(sV3V2x * ar) * atri.v1.w / frameWidth,(float)((-atri.v2.x * sV3V2y - atri.v2.y * sV3V2x) * ar) * atri.v1.w };
			atri.f2 = { (float)(sV1V3y * ar) * atri.v2.w / frameHeight, (float)(sV1V3x * ar) * atri.v2.w / frameWidth,(float)((-atri.v3.x * sV1V3y - atri.v3.y * sV1V3x) * ar) * atri.v2.w };


			ifloat3 edgeCoefs[3];
			atri.e1 = { (float)(sV2V1y)*frameHeight, (float)(sV2V1x)*frameWidth ,  (float)(atri.v2.x * atri.v1.y - atri.v1.x * atri.v2.y ) * frameHeight * frameWidth };
			atri.e2 = { (float)(sV3V2y)*frameHeight,  (float)(sV3V2x)*frameWidth ,  (float)(atri.v3.x * atri.v2.y - atri.v2.x * atri.v3.y ) * frameHeight * frameWidth };
			atri.e3 = { (float)(sV1V3y * frameHeight),  (float)(sV1V3x)*frameWidth ,  (float)(atri.v1.x * atri.v3.y - atri.v3.x * atri.v1.y ) * frameHeight * frameWidth };

			atri.originalPrimitive = primitiveId;
			irect2Df bbox;
			if (!devTriangleSimpleClip(atri.v1, atri.v2, atri.v3, bbox)) continue;
			if constexpr (CU_NOT_TILED_BINNER) {
				devExecuteBinner(idxSrc + i, atri, bbox, dRasterQueue, dRasterQueueCount, dCoverQueue, dCoverQueueCounts, deviceConstants);
			}
			dProposals[curIdx] = atri;
		}
		return  retCnt[clipOdd] - 2;
#undef ret
	}



	IFRIT_DEVICE void* devGetBufferAddress(char* dBuffer, TypeDescriptorEnum typeDesc, uint32_t element) {
		if (typeDesc == TypeDescriptorEnum::IFTP_FLOAT1) {
			return reinterpret_cast<float*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_FLOAT2) {
			return reinterpret_cast<ifloat2*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_FLOAT3) {
			return reinterpret_cast<ifloat3*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_FLOAT4) {
			return reinterpret_cast<ifloat4*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_INT1) {
			return reinterpret_cast<int*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_INT2) {
			return reinterpret_cast<iint2*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_INT3) {
			return reinterpret_cast<iint3*>(dBuffer) + element;
		}
		else if (typeDesc == TypeDescriptorEnum::IFTP_INT4) {
			return reinterpret_cast<iint4*>(dBuffer) + element;
		}
		else {
			return nullptr;
		}
	}

	
	IFRIT_DEVICE void devInterpolateVaryings(
		int id,
		VaryingStore** dVaryingBuffer,
		TypeDescriptorEnum* dVaryingTypeDescriptor,
		const int indices[3],
		const float barycentric[3],
		VaryingStore& dest
	) {
		auto va = dVaryingBuffer[id];
		auto varyingDescriptor = dVaryingTypeDescriptor[id];
		VaryingStore vd;
		if (varyingDescriptor == TypeDescriptorEnum::IFTP_FLOAT4) {
			vd.vf4 = { 0,0,0,0 };
			for (int j = 0; j < 3; j++) {
				auto vaf4 = va[indices[j]].vf4;
				vd.vf4.x += vaf4.x * barycentric[j];
				vd.vf4.y += vaf4.y * barycentric[j];
				vd.vf4.z += vaf4.z * barycentric[j];
				vd.vf4.w += vaf4.w * barycentric[j];
			}
			dest = vd;
		}
		else if (varyingDescriptor == TypeDescriptorEnum::IFTP_FLOAT3) {
			dest.vf3 = { 0,0,0 };
			for (int j = 0; j < 3; j++) {
				dest.vf3.x += va[indices[j]].vf3.x * barycentric[j];
				dest.vf3.y += va[indices[j]].vf3.y * barycentric[j];
				dest.vf3.z += va[indices[j]].vf3.z * barycentric[j];
			}

		}
		else if (varyingDescriptor == TypeDescriptorEnum::IFTP_FLOAT2) {
			dest.vf2 = { 0,0 };
			for (int j = 0; j < 3; j++) {
				dest.vf2.x += va[indices[j]].vf2.x * barycentric[j];
				dest.vf2.y += va[indices[j]].vf2.y * barycentric[j];
			}
		}
		else if (varyingDescriptor == TypeDescriptorEnum::IFTP_FLOAT1) {
			dest.vf = 0;
			for (int j = 0; j < 3; j++) {
				dest.vf += va[indices[j]].vf * barycentric[j];
			}
		}
	}



	IFRIT_DEVICE void devPixelShadingUnlocked(
		uint32_t pixelX,
		uint32_t pixelY,
		FragmentShader* fragmentShader,
		int* IFRIT_RESTRICT_CUDA dIndexBuffer,
		VaryingStore** IFRIT_RESTRICT_CUDA dVaryingBuffer,
		TypeDescriptorEnum* IFRIT_RESTRICT_CUDA dVaryingTypeDescriptor,
		AssembledTriangleProposalCUDA dAtp,
		ifloat4** IFRIT_RESTRICT_CUDA dColorBuffer,
		float* IFRIT_RESTRICT_CUDA dDepthBuffer,
		int frameBufferWidth,
		int frameBufferHeight,
		int vertexStride,
		int varyingCount
	) {
		VaryingStore interpolatedVaryings[CU_MAX_VARYINGS];
		ifloat4 colorOutputSingle;

		const AssembledTriangleProposalCUDA& atp = dAtp;
		ifloat4 pos[4];
		pos[0] = atp.v1;
		pos[1] = atp.v2;
		pos[2] = atp.v3;

		float pDx = 1.0f * pixelX;
		float pDy = 1.0f * pixelY;

		float bary[3];
		float depth[3];
		float interpolatedDepth;

		bary[0] = (atp.f1.x * pDx + atp.f1.y * pDy + atp.f1.z);
		bary[1] = (atp.f2.x * pDx + atp.f2.y * pDy + atp.f2.z);
		bary[2] = (atp.f3.x * pDx + atp.f3.y * pDy + atp.f3.z);
		interpolatedDepth = bary[0] * pos[0].z + bary[1] * pos[1].z + bary[2] * pos[2].z;
		float zCorr = 1.0f / (bary[0] + bary[1] + bary[2]);
		interpolatedDepth *= zCorr;
		const auto pixelPos = pixelY * frameBufferWidth + pixelX;
		auto depthRef = dDepthBuffer[pixelPos];
		if (interpolatedDepth <= depthRef) {
			float desiredBary[3];
			bary[0] *= zCorr;
			bary[1] *= zCorr;
			bary[2] *= zCorr;
			desiredBary[0] = bary[0] * atp.b1.x + bary[1] * atp.b2.x + bary[2] * atp.b3.x;
			desiredBary[1] = bary[0] * atp.b1.y + bary[1] * atp.b2.y + bary[2] * atp.b3.y;
			desiredBary[2] = bary[0] * atp.b1.z + bary[1] * atp.b2.z + bary[2] * atp.b3.z;
			auto addr = dIndexBuffer + atp.originalPrimitive * vertexStride;
			for (int k = 0; k < varyingCount; k++) {
				devInterpolateVaryings(k, dVaryingBuffer, dVaryingTypeDescriptor,addr, desiredBary, interpolatedVaryings[k]);
			}
			fragmentShader->execute(interpolatedVaryings, &colorOutputSingle);
			dColorBuffer[0][pixelPos] = colorOutputSingle;
			dDepthBuffer[pixelPos] = interpolatedDepth;
		}
	}

	IFRIT_DEVICE void devTilingRasterizationChildProcess(
		uint32_t tileIdX,
		uint32_t tileIdY,
		uint32_t invoId,
		uint32_t totalBound,
		AssembledTriangleProposalCUDA* dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueue,
		TileBinProposalCUDA* IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;

		auto globalInvocation = invoId;
		if (globalInvocation > totalBound)return;

		const auto tileId = tileIdY * CU_TILE_SIZE + tileIdX;
		const auto frameWidth = deviceConstants->frameBufferWidth;
		const auto frameHeight = deviceConstants->frameBufferHeight;

		const uint32_t pixelStX = frameWidth * tileIdX / CU_TILE_SIZE;
		const uint32_t pixelEdX = frameWidth * (tileIdX + 1) / CU_TILE_SIZE;
		const uint32_t pixelStY = frameHeight * tileIdY / CU_TILE_SIZE;
		const uint32_t pixelEdY = frameHeight * (tileIdY + 1) / CU_TILE_SIZE;
		const auto primitiveSrcId = dRasterQueue[globalInvocation];

		const auto& atri = dAssembledTriangles[primitiveSrcId];

		ifloat3 edgeCoefs[3];
		edgeCoefs[0] = atri.e1;
		edgeCoefs[1] = atri.e2;
		edgeCoefs[2] = atri.e3;

		int chosenCoordTR[3];
		int chosenCoordTA[3];
		devGetAcceptRejectCoords(edgeCoefs, chosenCoordTR, chosenCoordTA);

		auto curTileX = tileIdX * frameWidth / CU_TILE_SIZE;
		auto curTileY = tileIdY * frameHeight / CU_TILE_SIZE;
		auto curTileX2 = (tileIdX + 1) * frameWidth / CU_TILE_SIZE;
		auto curTileY2 = (tileIdY + 1) * frameHeight / CU_TILE_SIZE;
		auto curTileWid = curTileX2 - curTileX;
		auto curTileHei = curTileY2 - curTileY;

		const float dEps = CU_EPS * frameHeight * frameWidth;
		// Decomp into Sub Blocks
		for (int i = CU_SUBTILE_SIZE * CU_SUBTILE_SIZE - 1; i >= 0; --i) {
			int criteriaTR = 0;
			int criteriaTA = 0;

			auto subTileIX = i % CU_SUBTILE_SIZE;
			auto subTileIY = i / CU_SUBTILE_SIZE;
			auto subTileTX = (tileIdX * CU_SUBTILE_SIZE + subTileIX);
			auto subTileTY = (tileIdY * CU_SUBTILE_SIZE + subTileIY);

			const int wp = (CU_SUBTILE_SIZE * CU_TILE_SIZE);
			int subTilePixelX = curTileX + (curTileWid * subTileIX >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelY = curTileY + (curTileHei * subTileIY >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelX2 = curTileX + (curTileWid * (subTileIX + 1) >> CU_SUBTILE_SIZE_LOG);
			int subTilePixelY2 = curTileY + (curTileHei * (subTileIY + 1) >> CU_SUBTILE_SIZE_LOG);

			float subTileMinX = 1.0f * subTilePixelX;
			float subTileMinY = 1.0f * subTilePixelY;
			float subTileMaxX = 1.0f * (subTilePixelX2-1);
			float subTileMaxY = 1.0f * (subTilePixelY2-1);


			ifloat2 tileCoords[4];
			tileCoords[VLT] = { subTileMinX, subTileMinY };
			tileCoords[VLB] = { subTileMinX, subTileMaxY };
			tileCoords[VRB] = { subTileMaxX, subTileMaxY };
			tileCoords[VRT] = { subTileMaxX, subTileMinY };

			const float cmpf[3] = { dEps - edgeCoefs[0].z,dEps - edgeCoefs[1].z,dEps - edgeCoefs[2].z };
			for (int k = 0; k < 3; k++) {
				float criteriaTRLocal = edgeCoefs[k].x * tileCoords[chosenCoordTR[k]].x + edgeCoefs[k].y * tileCoords[chosenCoordTR[k]].y;
				float criteriaTALocal = edgeCoefs[k].x * tileCoords[chosenCoordTA[k]].x + edgeCoefs[k].y * tileCoords[chosenCoordTA[k]].y;
				criteriaTR += criteriaTRLocal < cmpf[k];
				criteriaTA += criteriaTALocal < cmpf[k];
			}

			if (criteriaTR != 3) {
				continue;
			}
			if (criteriaTA == 3) {
				TileBinProposalCUDA nprop;
				nprop.tileEnd = { (short)(subTilePixelX2 - 1),(short)(subTilePixelY2 - 1) };
				nprop.tile = { (short)subTilePixelX,(short)subTilePixelY };
				nprop.primId = primitiveSrcId;
				auto proposalInsIdx = atomicAdd(dCoverQueueCount, 1);
				dCoverQueue[proposalInsIdx] = nprop;
			}
			else {
				//Into Pixel level
				int wid = subTilePixelX2 - subTilePixelX;
				int hei = subTilePixelY2 - subTilePixelY;
				int tot = wid * hei;
				for (int i2 = tot - 1; i2 >= 0; i2--) {
					int dx = subTilePixelX + i2 % wid;
					int dy = subTilePixelY + i2 / wid;
					int accept = 0;
					for (int i = 0; i < 3; i++) {
						float criteria = edgeCoefs[i].x * dx + edgeCoefs[i].y * dy;
						accept += criteria < cmpf[i];
					}
					if (accept == 3) {
						TileBinProposalCUDA nprop;
						nprop.tileEnd = { (short)dx,(short)dy };
						nprop.tile = { (short)dx,(short)dy };
						nprop.primId = primitiveSrcId;
						auto proposalInsIdx = atomicAdd(dCoverQueueCount, 1);
						dCoverQueue[proposalInsIdx] = nprop;
					}
				}
			}
		}
	}


	// Kernel Implementations

	IFRIT_KERNEL void vertexProcessingKernel(
		VertexShader* vertexShader,
		uint32_t vertexCount,
		char* dVertexBuffer,
		TypeDescriptorEnum* dVertexTypeDescriptor,
		VaryingStore** dVaryingBuffer,
		TypeDescriptorEnum* dVaryingTypeDescriptor,
		ifloat4* dPosBuffer,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto globalInvoIdx = blockIdx.x * blockDim.x + threadIdx.x;
		if (globalInvoIdx >= vertexCount) return;
		const auto numAttrs = deviceConstants->attributeCount;
		const auto numVaryings = deviceConstants->varyingCount;

		const void* vertexInputPtrs[CU_MAX_ATTRIBUTES];
		VaryingStore* varyingOutputPtrs[CU_MAX_VARYINGS];

		int offsets[CU_MAX_ATTRIBUTES];
		int totalOffset = 0;
		for (int i = 0; i < numAttrs; i++) {
			int cof = 0;
			offsets[i] = totalOffset;
			if (dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT1) cof  = sizeof(float);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT2) cof = sizeof(ifloat2);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT3) cof = sizeof(ifloat3);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_FLOAT4)cof = sizeof(ifloat4);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT1) cof = sizeof(int);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT2) cof = sizeof(iint2);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT3) cof = sizeof(iint3);
			else if(dVertexTypeDescriptor[i] == TypeDescriptorEnum::IFTP_INT4) cof = sizeof(iint4);
			totalOffset += cof;
		}
		for (int i = 0; i < numAttrs; i++) {
			vertexInputPtrs[i] = globalInvoIdx * totalOffset + dVertexBuffer + offsets[i];
		}
		for (int i = 0; i < numVaryings; i++) {
			varyingOutputPtrs[i] = dVaryingBuffer[i] + globalInvoIdx;
		}
		vertexShader->execute(vertexInputPtrs, &dPosBuffer[globalInvoIdx], varyingOutputPtrs);
	}

	IFRIT_KERNEL void geometryProcessingKernel(
		ifloat4* dPosBuffer,
		int* dIndexBuffer,
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		uint32_t startingIndexId,
		uint32_t indexCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto globalInvoIdx = blockIdx.x * blockDim.x + threadIdx.x;
		if(globalInvoIdx >= indexCount / CU_TRIANGLE_STRIDE) return;

		const auto indexStart = globalInvoIdx * CU_TRIANGLE_STRIDE + startingIndexId;
		ifloat4 v1 = dPosBuffer[dIndexBuffer[indexStart]];
		ifloat4 v2 = dPosBuffer[dIndexBuffer[indexStart + 1]];
		ifloat4 v3 = dPosBuffer[dIndexBuffer[indexStart + 2]];
		if (deviceConstants->counterClockwise) {
			ifloat4 temp = v1;
			v1 = v3;
			v3 = temp;
		}
		
		const auto primId = globalInvoIdx + startingIndexId / CU_TRIANGLE_STRIDE;
		if (!devTriangleCull(v1, v2, v3)) {
			return;
		}
		devTriangleHomogeneousClip(primId, v1, v2, v3, dAssembledTriangles, dAssembledTriangleCount, 
			deviceConstants->frameBufferWidth, deviceConstants->frameBufferHeight, dRasterQueue, dRasterQueueCount,
			dCoverQueue, dCoverQueueCount, deviceConstants);
		
	}

	IFRIT_KERNEL void tilingBinnerKernel(
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		constexpr const int VLB = 0, VLT = 1, VRT = 2, VRB = 3;

		const auto tileX = blockIdx.x;
		const auto tileY = blockIdx.y;
		const auto threadX = threadIdx.x;
		const auto numThreads = blockDim.x;
		const auto totalTriangles = *dAssembledTriangleCount;
		const auto frameBufferWidth = deviceConstants->frameBufferWidth;
		const auto frameBufferHeight = deviceConstants->frameBufferHeight;

		auto curTileX = tileX * frameBufferWidth / CU_TILE_SIZE;
		auto curTileY = tileY * frameBufferHeight / CU_TILE_SIZE;
		auto curTileX2 = (tileX + 1) * frameBufferWidth / CU_TILE_SIZE;
		auto curTileY2 = (tileY + 1) * frameBufferHeight / CU_TILE_SIZE;

		auto ctx1 = 1.0f * curTileX;
		auto ctx2 = 1.0f * (curTileX2 - 1);
		auto cty1 = 1.0f * curTileY;
		auto cty2 = 1.0f * (curTileY2 - 1);

		ifloat2 tileCoords[4];
		tileCoords[VLT] = { ctx1, cty1 };
		tileCoords[VLB] = { ctx1, cty2 };
		tileCoords[VRB] = { ctx2, cty2 };
		tileCoords[VRT] = { ctx2, cty1 };

		for (int i = threadX; i < totalTriangles; i+=numThreads) {
			const auto& atp = dAssembledTriangles[i];
			ifloat3 edgeCoefs[3];
			edgeCoefs[0] = atp.e1;
			edgeCoefs[1] = atp.e2;
			edgeCoefs[2] = atp.e3;

			int chosenCoordTR[3];
			int chosenCoordTA[3];
			devGetAcceptRejectCoords(edgeCoefs, chosenCoordTR, chosenCoordTA);

			int criteriaTR = 0;
			int criteriaTA = 0;
			for (int i = 0; i < 3; i++) {
				float criteriaTRLocal = edgeCoefs[i].x * tileCoords[chosenCoordTR[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTR[i]].y + edgeCoefs[i].z;
				float criteriaTALocal = edgeCoefs[i].x * tileCoords[chosenCoordTA[i]].x + edgeCoefs[i].y * tileCoords[chosenCoordTA[i]].y + edgeCoefs[i].z;
				if (criteriaTRLocal < -CU_EPS) criteriaTR += 1;
				if (criteriaTALocal < CU_EPS) criteriaTA += 1;
			}
			if (criteriaTR != 3) {
				continue;
			}
			auto workerId = threadIdx.x;
			auto tileId = tileY * CU_TILE_SIZE + tileX;
			if (criteriaTA == 3) {
				TileBinProposalCUDA proposal;
				proposal.tileEnd = { (short)(curTileX2 - 1),(short)(curTileY2 - 1) };
				proposal.primId = i;
				proposal.tile = { (short)curTileX,(short)curTileY };
				auto proposalId = atomicAdd(&dCoverQueueCount[tileId], 1);
				dCoverQueue[tileId][proposalId] = proposal;
			}
			else {
				auto proposalId = atomicAdd(&dRasterQueueCount[tileId], 1);
				dRasterQueue[tileId][proposalId] = i;
			}
		}
	}


	IFRIT_KERNEL void tilingRasterizationKernel(
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAssembledTriangles,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembledTriangleCount,
		uint32_t** IFRIT_RESTRICT_CUDA dRasterQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		TileRasterDeviceConstants* deviceConstants
	) {
		const auto tileIdxX = blockIdx.x ;
		const auto tileIdxY = blockIdx.y;
		const auto threadX = threadIdx.x;
		const auto blockX = blockDim.x;
		const auto tileId = tileIdxY * CU_TILE_SIZE+ tileIdxX;
		IFRIT_SHARED uint32_t sdAtomicCounter[1];
		IFRIT_SHARED uint32_t sdRastCandidates;
		if (threadX == 0) {
			sdAtomicCounter[0] = dCoverQueueCount[tileId];
			sdRastCandidates = dRasterQueueCount[tileId];
		}
		__syncthreads();
		const auto dRaster = dRasterQueue[tileId];
		const auto dCover = dCoverQueue[tileId];
		for (int i = threadX; i < sdRastCandidates; i+= blockX) {
			devTilingRasterizationChildProcess(tileIdxX, tileIdxY, i, sdRastCandidates, dAssembledTriangles,
				dRaster, dCover, sdAtomicCounter, deviceConstants);
		}
		__syncthreads();
		if (threadX == 0) {
			dCoverQueueCount[tileId] = sdAtomicCounter[0];
		}
	}


	IFRIT_KERNEL void fragmentShadingKernelPerTile(
		FragmentShader*  fragmentShader,
		int* IFRIT_RESTRICT_CUDA dIndexBuffer,
		VaryingStore** IFRIT_RESTRICT_CUDA dVaryingBuffer,
		TypeDescriptorEnum* IFRIT_RESTRICT_CUDA dVaryingTypeDescriptor,
		TileBinProposalCUDA** IFRIT_RESTRICT_CUDA dCoverQueue,
		uint32_t* IFRIT_RESTRICT_CUDA dCoverQueueCount,
		uint32_t* IFRIT_RESTRICT_CUDA dRasterQueueCount,
		uint32_t* IFRIT_RESTRICT_CUDA dAssembleTriangleCounter,
		AssembledTriangleProposalCUDA* IFRIT_RESTRICT_CUDA dAtp,
		ifloat4** IFRIT_RESTRICT_CUDA dColorBuffer,
		float* IFRIT_RESTRICT_CUDA dDepthBuffer,
		TileRasterDeviceConstants* deviceConstants
	) {
		uint32_t tileX = blockIdx.x;
		uint32_t tileY = blockIdx.y;

		uint32_t tileId = tileY * CU_TILE_SIZE + tileX;
		const auto frameWidth = deviceConstants->frameBufferWidth;
		const auto frameHeight = deviceConstants->frameBufferHeight;
		const auto candidates = dCoverQueueCount[tileId];
		constexpr auto vertexStride = CU_TRIANGLE_STRIDE;
		const auto varyingCount = deviceConstants->varyingCount;

		const int threadX = threadIdx.x;
		const int threadY = threadIdx.y;
		constexpr auto bds = CU_FRAGMENT_SHADING_THREADS_PER_TILE_X * CU_FRAGMENT_SHADING_THREADS_PER_TILE_Y;
		const auto threadId = threadY * bds + threadX;

		constexpr int blockX = CU_FRAGMENT_SHADING_THREADS_PER_TILE_X;
		constexpr int blockY = CU_FRAGMENT_SHADING_THREADS_PER_TILE_Y;
		

		IFRIT_SHARED TileBinProposalCUDA* sdCoverQueueSrc;
		IFRIT_SHARED TypeDescriptorEnum sdVaryingTypeDescriptor[CU_MAX_VARYINGS];
		IFRIT_SHARED TileBinProposalCUDA sdCoverQueue[CU_FRAGMENT_CANDPROC_PER_LOOP];
		if (threadId == 0) {
			sdCoverQueueSrc = dCoverQueue[tileId];
		}
		if(threadId<varyingCount) {
			sdVaryingTypeDescriptor[threadId] = dVaryingTypeDescriptor[threadId];
		}

		for (int k = 0; k < candidates; k += CU_FRAGMENT_CANDPROC_PER_LOOP) {
			auto mf = min(candidates - k, CU_FRAGMENT_CANDPROC_PER_LOOP);
			for (int i = threadId; i < mf; i += CU_FRAGMENT_CANDPROC_PER_LOOP_PER_THREAD) {
				sdCoverQueue[i] = sdCoverQueueSrc[i + k];
			}
			__syncthreads();

			for (int i = mf-1; i >=0 ; i--) {
				const auto proposal = sdCoverQueue[i];
				const auto atp = dAtp[proposal.primId];
				const auto startX = proposal.tile.x;
				const auto startY = proposal.tile.y;
				const auto endX = proposal.tileEnd.x;
				const auto endY = proposal.tileEnd.y;

#define FIND_SMALLEST(a,b,q) ((a) * (((q) - (b) + (a) - 1) / (a)) + (b))
#define FIND_LARGEST(a,b,q) (((q) - (b)) / (a) * (a) + (b))
				const auto startXw = max(0,FIND_SMALLEST(blockX, threadX, startX));
				const auto endXw = min(frameWidth - 1, FIND_LARGEST(blockX, threadX, endX));
				const auto startYw = max(0,FIND_SMALLEST(blockY, threadY, startY));
				const auto endYw = min(frameHeight - 1, FIND_LARGEST(blockY, threadY, endY));
#undef FIND_SMALLEST
#undef FIND_LARGEST

				for (int pixelX = startXw; pixelX <= endXw; pixelX += blockX) {
					for (int pixelY = startYw; pixelY <= endYw; pixelY += blockY) {
						devPixelShadingUnlocked(pixelX, pixelY, fragmentShader, dIndexBuffer, dVaryingBuffer,
							sdVaryingTypeDescriptor, atp, dColorBuffer, dDepthBuffer,
							frameWidth, frameHeight, vertexStride, varyingCount);
					}
				}
				
			}
			__syncthreads();
		}
		
		//Reset kernels
		if (threadX == 0) {
			dCoverQueueCount[tileId] = 0;
			dRasterQueueCount[tileId] = 0;
			dAssembleTriangleCounter[0] = 0;
		}

	}


	IFRIT_KERNEL void imageResetFloat32Kernel(
		float* dBuffer,
		uint32_t imageX,
		uint32_t imageY,
		uint32_t channels,
		float value
	) {
		const auto invoX = blockIdx.x * blockDim.x + threadIdx.x;
		const auto invoY = blockIdx.y * blockDim.y + threadIdx.y;
		if (invoX >= imageX || invoY >= imageY) {
			return;
		}
		for(int i=0;i<channels;i++) {
			dBuffer[(invoY * imageX + invoX) * channels + i] = value;
		}
	}

	IFRIT_KERNEL void imageResetInt32Kernel(
		int* dBuffer,
		uint32_t imageX,
		uint32_t imageY,
		uint32_t channels,
		int value
	) {
		const auto invoX = blockIdx.x * blockDim.x + threadIdx.x;
		const auto invoY = blockIdx.y * blockDim.y + threadIdx.y;
		if (invoX >= imageX || invoY >= imageY) {
			return;
		}
		for (int i = 0; i < channels; i++) {
			dBuffer[(invoY * imageX + invoX) * channels + i] = value;
		}
	}

	IFRIT_KERNEL void resetKernel(
		uint32_t* count,
		uint32_t size
	) {
		const auto globalInvocation = blockIdx.x * blockDim.x + threadIdx.x;
		if (globalInvocation >= size) {
			return;
		}
		count[globalInvocation] = 0;
	}

	IFRIT_KERNEL void testingKernel() {
		printf("Hello World\n");
	}
}


namespace  Ifrit::Engine::TileRaster::CUDA::Invocation {


	template<typename T>
	__global__ void kernFixVTable(T* devicePtr) {
		T temp(*devicePtr);
		memcpy(devicePtr, &temp, sizeof(T));
	}

	template<typename T>
	__host__ T* hostGetDeviceObjectCopy(T* hostObject) {
		T* deviceHandle;
		hipMalloc(&deviceHandle, sizeof(T));
		hipMemcpy(deviceHandle, hostObject, sizeof(T), hipMemcpyHostToDevice);
		printf("Copying object to CUDA, %lld,%d\n", deviceHandle, 1);
		kernFixVTable<T> CU_KARG2(1, 1)(deviceHandle);
		hipDeviceSynchronize();
		printf("Cuda Addr=%lld\n", deviceHandle);
		printf("Object copied to CUDA\n");
		return deviceHandle;
	}

	template<class T>
	T* copyShaderToDevice(T* x) {
		return hostGetDeviceObjectCopy<T>(x);
	}

	void testingKernelWrapper() {
		Impl::testingKernel CU_KARG2(4,4) ();
		hipDeviceSynchronize();
	}
	char* deviceMalloc(uint32_t size) {
		char* ptr;
		hipMalloc(&ptr, size);
		return ptr;
	
	}
	void deviceFree(char* ptr) {
		hipFree(ptr);
	}


	int* getIndexBufferDeviceAddr(const int* hIndexBuffer, uint32_t indexBufferSize, int* dOldIndexBuffer) {
		if(dOldIndexBuffer != nullptr) {
			hipFree(dOldIndexBuffer);
		}
		int* dIndexBuffer;
		hipMalloc(&dIndexBuffer, indexBufferSize * sizeof(int));
		hipMemcpy(dIndexBuffer, hIndexBuffer, indexBufferSize * sizeof(int), hipMemcpyHostToDevice);
		return dIndexBuffer;
	}
	char* getVertexBufferDeviceAddr(const char* hVertexBuffer, uint32_t bufferSize, char* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		char* dBuffer;
		hipMalloc(&dBuffer, bufferSize);
		hipMemcpy(dBuffer, hVertexBuffer, bufferSize, hipMemcpyHostToDevice);
		return dBuffer;
	
	}
	TypeDescriptorEnum* getTypeDescriptorDeviceAddr(const TypeDescriptorEnum* hBuffer, uint32_t bufferSize, TypeDescriptorEnum* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		TypeDescriptorEnum* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(TypeDescriptorEnum));
		hipMemcpy(dBuffer, hBuffer, bufferSize * sizeof(TypeDescriptorEnum), hipMemcpyHostToDevice);
		return dBuffer;
	
	}
	float* getDepthBufferDeviceAddr(uint32_t bufferSize, float* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		float* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(float));
		return dBuffer;
	}
	ifloat4* getPositionBufferDeviceAddr(uint32_t bufferSize, ifloat4* dOldBuffer) {
		if(dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		ifloat4* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(ifloat4));
		return dBuffer;
	
	}
	int* getShadingLockDeviceAddr(uint32_t bufferSize, int* dOldBuffer) {
		if (dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		int* dBuffer;
		hipMalloc(&dBuffer, bufferSize * sizeof(int));
		return dBuffer;
	}
	void getColorBufferDeviceAddr(
		const std::vector<ifloat4*>& hColorBuffer,
		std::vector<ifloat4*>& dhColorBuffer,
		ifloat4**& dColorBuffer,
		uint32_t bufferSize,
		std::vector<ifloat4*>& dhOldColorBuffer,
		ifloat4** dOldBuffer
	) {
		//TODO: POTENTIAL BUGS & CUDA DEVICE MEMORY LEAK
		if (dOldBuffer != nullptr) {
			hipFree(dOldBuffer);
		}
		dhColorBuffer.resize(hColorBuffer.size());

		hipMalloc(&dColorBuffer, hColorBuffer.size() * sizeof(ifloat4*));
		for (int i = 0; i < hColorBuffer.size(); i++) {
			hipMalloc(&dhColorBuffer[i], bufferSize * sizeof(ifloat4));
			hipMemcpy(dhColorBuffer[i], hColorBuffer[i], bufferSize * sizeof(ifloat4), hipMemcpyHostToDevice);
		}
		hipMemcpy(dColorBuffer, dhColorBuffer.data(), hColorBuffer.size() * sizeof(ifloat4*), hipMemcpyHostToDevice);
	

	}

	void invokeCudaRendering(
		char* dVertexBuffer,
		TypeDescriptorEnum* dVertexTypeDescriptor,
		TypeDescriptorEnum* dVaryingTypeDescriptor,
		int* dIndexBuffer,
		int* dShaderLockBuffer,
		VertexShader* dVertexShader,
		FragmentShader* dFragmentShader,
		ifloat4** dColorBuffer,
		ifloat4** dHostColorBuffer,
		ifloat4** hColorBuffer,
		uint32_t dHostColorBufferSize,
		float* dDepthBuffer,
		ifloat4* dPositionBuffer,
		TileRasterDeviceConstants* deviceConstants,
		TileRasterDeviceContext* deviceContext,
		bool doubleBuffering,
		ifloat4** dLastColorBuffer
	) {
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		hipMemcpy(deviceContext->dDeviceConstants, deviceConstants, sizeof(TileRasterDeviceConstants), hipMemcpyHostToDevice);

		// Stream Preparation
		static int initFlag = 0;
		static hipStream_t copyStream, computeStream;
		static hipEvent_t  copyStart, copyEnd;
		if (initFlag == 0) {
			initFlag = 1;
			hipStreamCreate(&copyStream);
			hipStreamCreate(&computeStream);
			hipEventCreate(&copyStart);
			hipEventCreate(&copyEnd);
		}
		
		// Compute
		std::chrono::steady_clock::time_point end1 = std::chrono::steady_clock::now();

		constexpr int dispatchThreadsX = 8;
		constexpr int dispatchThreadsY = 8;
		int dispatchBlocksX = (deviceConstants->frameBufferWidth / dispatchThreadsX) + ((deviceConstants->frameBufferWidth % dispatchThreadsX) != 0);
		int dispatchBlocksY = (deviceConstants->frameBufferHeight / dispatchThreadsY) + ((deviceConstants->frameBufferHeight % dispatchThreadsY) != 0);

		Impl::imageResetFloat32Kernel CU_KARG4(dim3(dispatchBlocksX, dispatchBlocksY), dim3(dispatchThreadsX, dispatchThreadsY), 0, computeStream)(
			dDepthBuffer, deviceConstants->frameBufferWidth, deviceConstants->frameBufferHeight, 1, 255.0f
		);
		for (int i = 0; i < dHostColorBufferSize; i++) {
			Impl::imageResetFloat32Kernel CU_KARG4(dim3(dispatchBlocksX, dispatchBlocksY), dim3(dispatchThreadsX, dispatchThreadsY), 0, computeStream)(
				(float*)dHostColorBuffer[i], deviceConstants->frameBufferWidth, deviceConstants->frameBufferHeight, 4, 0.0f
			);
		}
		int vertexExecutionBlocks = (deviceConstants->vertexCount / CU_VERTEX_PROCESSING_THREADS) + ((deviceConstants->vertexCount % CU_VERTEX_PROCESSING_THREADS) != 0);
		Impl::vertexProcessingKernel CU_KARG4(vertexExecutionBlocks, CU_VERTEX_PROCESSING_THREADS, 0, computeStream)(
			dVertexShader, deviceConstants->vertexCount, dVertexBuffer, dVertexTypeDescriptor,
			deviceContext->dVaryingBuffer, dVaryingTypeDescriptor, dPositionBuffer, deviceContext->dDeviceConstants
		);
		
		constexpr int totalTiles = CU_TILE_SIZE * CU_TILE_SIZE;
		Impl::resetKernel CU_KARG4(1, 1, 0, computeStream)(deviceContext->dAssembledTrianglesCounter2, totalTiles);
		Impl::resetKernel CU_KARG4(CU_TILE_SIZE, CU_TILE_SIZE, 0, computeStream)(deviceContext->dRasterQueueCounter, totalTiles);
		Impl::resetKernel CU_KARG4(CU_TILE_SIZE, CU_TILE_SIZE, 0, computeStream)(deviceContext->dCoverQueueCounter, totalTiles);
		int cntw = 0;
		for (int i = 0; i < deviceConstants->totalIndexCount; i += CU_SINGLE_TIME_TRIANGLE * 3) {
			cntw++;
			auto indexCount = std::min(CU_SINGLE_TIME_TRIANGLE * 3, deviceConstants->totalIndexCount - i);
			int geometryExecutionBlocks = (indexCount / CU_TRIANGLE_STRIDE / CU_GEOMETRY_PROCESSING_THREADS) + ((indexCount / CU_TRIANGLE_STRIDE % CU_GEOMETRY_PROCESSING_THREADS) != 0);
			Impl::geometryProcessingKernel CU_KARG4(geometryExecutionBlocks, CU_GEOMETRY_PROCESSING_THREADS, 0, computeStream)(
				dPositionBuffer, dIndexBuffer, deviceContext->dAssembledTriangles2, deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dRasterQueue,deviceContext->dRasterQueueCounter, deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter,i, indexCount,
				deviceContext->dDeviceConstants
				);
			if constexpr (CU_TILED_BINNER) {
				Impl::tilingBinnerKernel CU_KARG4(dim3(CU_TILE_SIZE, CU_TILE_SIZE, 1), dim3(CU_RASTERIZATION_THREADS_PER_TILE, 1, 1), 0, computeStream)(
					deviceContext->dAssembledTriangles2, deviceContext->dAssembledTrianglesCounter2,
					deviceContext->dRasterQueue, deviceContext->dRasterQueueCounter, deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter, deviceContext->dDeviceConstants
					);
			}
			Impl::tilingRasterizationKernel CU_KARG4(dim3(CU_TILE_SIZE, CU_TILE_SIZE, 1), dim3(CU_RASTERIZATION_THREADS_PER_TILE, 1, 1), 0, computeStream)(
				deviceContext->dAssembledTriangles2, deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dRasterQueue, deviceContext->dRasterQueueCounter, deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter, deviceContext->dDeviceConstants
				);
			Impl::fragmentShadingKernelPerTile CU_KARG4(dim3(CU_TILE_SIZE, CU_TILE_SIZE, 1), dim3(CU_FRAGMENT_SHADING_THREADS_PER_TILE_X, CU_FRAGMENT_SHADING_THREADS_PER_TILE_Y, 1), 0, computeStream) (
				dFragmentShader, dIndexBuffer, deviceContext->dVaryingBuffer, dVaryingTypeDescriptor,
				deviceContext->dCoverQueue2, deviceContext->dCoverQueueCounter, deviceContext->dRasterQueueCounter,deviceContext->dAssembledTrianglesCounter2,
				deviceContext->dAssembledTriangles2, dColorBuffer, dDepthBuffer, deviceContext->dDeviceConstants
				);
		}
		if (!doubleBuffering) {
			hipDeviceSynchronize();
		}

		// Memory Copy
		std::chrono::steady_clock::time_point end2 = std::chrono::steady_clock::now();
		if (doubleBuffering) {
			for (int i = 0; i < dHostColorBufferSize; i++) {
				hipMemcpyAsync(hColorBuffer[i], dLastColorBuffer[i], deviceConstants->frameBufferWidth * deviceConstants->frameBufferHeight * sizeof(ifloat4), hipMemcpyDeviceToHost, copyStream);
			}
		}
		hipStreamSynchronize(computeStream);
		if (doubleBuffering) {
			hipStreamSynchronize(copyStream);
		}

		if (!doubleBuffering) {
			for (int i = 0; i < dHostColorBufferSize; i++) {
				hipMemcpy(hColorBuffer[i], dHostColorBuffer[i], deviceConstants->frameBufferWidth * deviceConstants->frameBufferHeight * sizeof(ifloat4), hipMemcpyDeviceToHost);
			}
		}
		std::chrono::steady_clock::time_point end3 = std::chrono::steady_clock::now();

		// End of rendering
		auto memcpyTimes = std::chrono::duration_cast<std::chrono::microseconds>(end1 - begin).count();
		auto computeTimes = std::chrono::duration_cast<std::chrono::microseconds>(end2 - end1).count();
		auto copybackTimes = std::chrono::duration_cast<std::chrono::microseconds>(end3 - end2).count();

		printf("Memcpy,Compute,Copyback,Counter: %lld,%lld,%lld,%d\n", memcpyTimes, computeTimes, copybackTimes,cntw);
	}
}